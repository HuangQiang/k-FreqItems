#include "hip/hip_runtime.h"
#include "util.cuh"

namespace clustering {

int g_k               = -1;         // global param: #clusters
f32 g_mae             = -1.0f;      // global param: mean absolute error
f32 g_mse             = -1.0f;      // global param: mean square   error
f64 g_tot_wc_time     = -1.0;       // global param: total wall clock time (s)
f64 g_tot_cpu_time    = -1.0;       // global param: total cpu time (s)

int g_num_buckets     = -1;         // global param: local number of buckets
int g_tot_buckets     = -1;         // global param: total number of buckets
int g_tot_bins        = -1;         // global param: total number of bins
int g_tot_seeds       = -1;         // global param: total number of seeds
f64 g_phase1_wc_time  = -1.0;       // global param: phase 1 wall clock time (s)
f64 g_phase2_wc_time  = -1.0;       // global param: phase 2 wall clock time (s)
f64 g_phase3_wc_time  = -1.0;       // global param: phase 3 wall clock time (s)
f64 g_eval_wc_time    = -1.0;       // global param: eval wall clock time (s)
f64 g_eval_cpu_time   = -1.0;       // global param: eval cpu time (s)
f64 g_silk_wc_time    = -1.0;       // global param: silk wall clock time (s)
f64 g_silk_cpu_time   = -1.0;       // global param: silk cpu time (s)

int g_iter            = -1;         // global param: #iterations
f64 g_init_wc_time    = -1.0;       // global param: init wall clock time (s)
f64 g_init_cpu_time   = -1.0;       // global param: init cpu time (s)
f64 g_iter_wc_time    = -1.0;       // global param: iter wall clock time (s)
f64 g_iter_cpu_time   = -1.0;       // global param: iter cpu time (s)
f64 g_kfreqitems_wc_time  = -1.0;   // global param: k-freqitems wall clock time (s)
f64 g_kfreqitems_cpu_time = -1.0;   // global param: k-freqitems cpu time (s)

// -----------------------------------------------------------------------------
void create_dir(                    // create dir if the path does not exist
    char *path)                         // input path
{
    int len = (int) strlen(path);
    for (int i = 0; i < len; ++i) {
        if (path[i] != '/') continue;
        
        char ch = path[i+1]; path[i+1] = '\0';
        if (access(path, F_OK) != 0) { // create the directory if not exist
            if (mkdir(path, 0755) != 0) {
                printf("Could not create %s\n", path); exit(1);
            }
        }
        path[i+1] = ch;
    }
}

// -----------------------------------------------------------------------------
void init_mpi_comm(                 // initialize mpi communication
    MPI_INFO &mpi_info)                 // mpi_info (return)
{
    MPI_Init(nullptr, nullptr);
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_info.rank_);
    MPI_Get_processor_name(mpi_info.name_, &mpi_info.name_len_);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_info.size_);

    printf("-----------------------------------------------------------------\n"
        "Rank #%d: Initialize MPI with %s; Comunication starts\n\n", 
        mpi_info.rank_, mpi_info.name_);
}

// -----------------------------------------------------------------------------
void finalize_mpi_comm(             // finalize mpi communication
    const MPI_INFO &mpi_info)           // mpi_info
{
    MPI_Finalize();

    printf("Rank #%d: Finalize MPI with %s; Communication ends\n" 
        "-----------------------------------------------------------------\n\n",
        mpi_info.rank_, mpi_info.name_);
}

// -----------------------------------------------------------------------------
void copy_pos(                      // copy a partial datapos to another pos
    int   n,                            // length of partial datapos
    const u64* datapos,                 // partial datapos
    u64   *pos)                         // another pos (return)
{
    u64 start = datapos[0];
    for (int j = 0; j < n; ++j) pos[j] = datapos[j] - start;
}

// -----------------------------------------------------------------------------
void all_buff_to_cand_list(         // convert all buffers into cand list
    int   n,                            // length of all buffers
    const int *all_buff,                // all buffers
    std::unordered_map<int, std::vector<std::pair<int, int> > > &cand_list)
{
    int i = 0;
    while (i < n) {
        int   key = all_buff[i++];
        int   len = all_buff[i++];
        const int *cand = &all_buff[i]; i += len;
        const int *freq = &all_buff[i]; i += len;
        
        // Note: the value of cand_list[key] may be duplicated
        std::vector<std::pair<int, int> > &tmp = cand_list[key];
        tmp.reserve(tmp.size() + len);
        for (int j = 0; j < len; ++j) {
            tmp.push_back(std::make_pair(cand[j], freq[j]));
        }
    }
    assert(i == n);
}

// -----------------------------------------------------------------------------
void all_buff_to_cand_list(         // convert all buffers into cand list
    int   n,                            // length of all buffers
    const int *all_buff,                // all buffers
    std::unordered_map<int, std::vector<std::pair<int, int> > > &cand_list,
    std::unordered_map<int, int> &cand_cnt)
{
    int i = 0;
    while (i < n) {
        int   key  = all_buff[i++]; // get bin key
        int   num  = all_buff[i++]; // get bin num
        int   freq = all_buff[i++]; // get max freq
        int   len  = all_buff[i++]; // get number of candidates
        const int *cand = &all_buff[i]; i += len;
        
        // Note: the value of cand_list[key] may be duplicated
        std::vector<std::pair<int, int> > &tmp = cand_list[key];
        if (cand_cnt[key] < 0) continue;
        
        int new_size = tmp.size() + len;
        if (new_size > CAND_LEN_SIZE) {
            // printf("new_size=%d, cand_cnt=%d\n", new_size, cand_cnt[key]);
            cand_cnt[key] = -1;
        }
        else {
            tmp.reserve(new_size);
            for (int j = 0; j < len; ++j) {
                tmp.push_back(std::make_pair(cand[j], freq));
            }
            cand_cnt[key] += num; // update bucket counter
        }
    }
}

// -----------------------------------------------------------------------------
u64 get_total_coords(               // get total number of coordinates
    int size,                           // number of MPIs (size)
    u64 num_coords)                     // number of coordinates (local)
{
    // single-thread case
    if (size == 1) return num_coords;
    
    // multi-thread case
    u64 *r_num_coords = new u64[size];
    memset(r_num_coords, 0UL, sizeof(u64)*size); // init 0

    // broadcast num_coords to different threads
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Allgather(&num_coords, 1, MPI_UINT64_T, r_num_coords, 1, MPI_UINT64_T, 
        MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);

    // sum up the r_num_coords array
    u64 tot_coords = 0UL;
    for (int i = 0; i < size; ++i) tot_coords += r_num_coords[i];
    delete[] r_num_coords;
    
    return tot_coords;
}

// -----------------------------------------------------------------------------
int get_total_buckets(              // get total number of buckets
    int size,                           // number of MPIs (size)
    int num_buckets)                    // number of buckets (local)
{
    // single-thread case: g_tot_buckets == g_num_buckets
    if (size == 1) return num_buckets;
    
    // multi-thread case: broadcast num_buckets to different threads
    int *r_num_buckets = new int[size];
    memset(r_num_buckets, 0, sizeof(int)*size); // init 0

    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Allgather(&num_buckets, 1, MPI_INT, r_num_buckets, 1, MPI_INT, 
        MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);

    // sum up the r_num_buckets array
    u64 tot_buckets = 0UL;
    for (int i = 0; i < size; ++i) tot_buckets += r_num_buckets[i];
    delete[] r_num_buckets;
    
    if (tot_buckets > MAX_INT) exit(1); // only support range int
    return tot_buckets;
}

// -----------------------------------------------------------------------------
int get_max_round(                  // broadcast num_buffer to get max round
    int size,                           // number of MPIs (size)
    int num_buffer)                     // number of buffer (local)
{
    // single-thread case
    if (size == 1) return num_buffer + 1;
    
    // multi-thread case
    int *r_num_buffer = new int[size];
    for (int i = 0; i < size; ++i) r_num_buffer[i] = 0;

    // broadcast num_buffer to different threads
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Allgather(&num_buffer, 1, MPI_INT, r_num_buffer, 1, MPI_INT, 
        MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);

    // get max_round from the array r_num_buffer
    int max_round = r_num_buffer[0];
    for (int i = 1; i < size; ++i) {
        if (r_num_buffer[i] > max_round) max_round = r_num_buffer[i];
    }
    delete[] r_num_buffer;
    
    return max_round + 1; // start from 1
}

// -----------------------------------------------------------------------------
int gather_all_buffers(             // gather buffers from diff threads to root
    int   size,                         // number of MPIs (size)
    const std::vector<int> &buffer,     // buffer in local
    int   *all_buff)                    // all buffers at root (return)
{
    // gather the length of different buffers from different threads
    int len = (int) buffer.size(); // get the length of this buffer
    int *rlen = new int[size];
    for (int i = 0; i < size; ++i) rlen[i] = 0;
    
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Gather(&len, 1, MPI_INT, rlen, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    // get the total length of buffer (tlen) from all threads
    int tlen = 0;
    for (int i = 0; i < size; ++i) tlen += rlen[i];
    assert(tlen <= MAX_BUFF_SIZE);
    
    // init displacements to gather buffers from different threads to root
    int *displs = new int[size];
    displs[0] = 0;
    for (int i = 1; i < size; ++i) displs[i] = displs[i-1] + rlen[i-1];

    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Gatherv(buffer.data(), len, MPI_INT, all_buff, rlen, displs, MPI_INT, 
        0, MPI_COMM_WORLD); // 0 is the root location
    MPI_Barrier(MPI_COMM_WORLD);
    
    delete[] rlen;
    delete[] displs;
    
    return tlen;
}

// -----------------------------------------------------------------------------
void broadcast_set_and_pos(         // broadcast set and pos to other threads
    int rank,                           // MPI rank
    int size,                           // number of MPIs (size)
    std::vector<int> &binset,           // bin set (return)
    std::vector<u64> &binpos)           // bin position (return)
{
    // broadcast binpos from root to other threads
    u64 binpos_size = binpos.size();
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(&binpos_size, 1, MPI_UINT64_T, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    if (rank > 0) binpos.resize(binpos_size);
    
    assert(binpos_size <= MAX_INT);
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(binpos.data(), binpos_size, MPI_UINT64_T, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    // broadcast binset from root to other threads
    u64 binset_size = binset.size();
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(&binset_size, 1, MPI_UINT64_T, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    if (rank > 0) binset.resize(binset_size);
    
    int batch = MAX_BUFF_SIZE;
    for (size_t i = 0; i < binset_size; i += batch) {
        if (i+batch > binset_size) batch = binset_size-i;

        MPI_Barrier(MPI_COMM_WORLD);
        MPI_Bcast(binset.data()+i, batch, MPI_INT, 0, MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);
    }
}

// -----------------------------------------------------------------------------
u32 uniform_u32(                    // gen a random variable from uniform u32
    u32 min,                            // min value
    u32 max)                            // max value
{
    u32 r = 0U;
    if (RAND_MAX >= max - min) {
        r = min + (u32) ((max-min+1.0)*rand() / (RAND_MAX+1.0));
    }
    else {
        r = min + (u32) ((max-min+1.0) * 
            ((u64) rand() * ((u64) RAND_MAX+1.0) + (u64) rand()) / 
            ((u64) RAND_MAX*((u64) RAND_MAX+1.0) + (u64) RAND_MAX+1.0));
    }
    assert(r >= min && r <= max);
    return r; 
}

// -----------------------------------------------------------------------------
float uniform(                      // gen a random variable from uniform distr.
    float start,                        // start position
    float end)                          // end position
{
    assert(start < end);
    return start + ((end-start)*rand() / (float) RAND_MAX);
}

// -----------------------------------------------------------------------------
float gaussian()                    // gen a random variable from N(0,1)
{
    float x1 = -1.0f;
    do {
        x1 = uniform(0.0f, 1.0f);
    } while (x1 < FLOAT_ERROR); // cannot take log 0 for x1
    
    float x2 = uniform(0.0f, 1.0f);
    return sqrt(-2.0f*log(x1)) * cos(2.0f*PI*x2);
}

// -----------------------------------------------------------------------------
float cauchy()                      // gen a random variable from Cauchy(1,0)
{
    float x = gaussian();
    float y = gaussian();
    if (fabs(y) < FLOAT_ERROR) y = FLOAT_ERROR;
    
    return x / y;
}

// -----------------------------------------------------------------------------
void syn_hash_params(               // synchronize hash parameters
    int   len_s,                        // length of random projection
    int   len_p,                        // length of random shift
    float *shift,                       // random shift (return)
    float *proj)                        // random projection (return)
{
    // C int      --> MPI_INT
    // C float    --> MPI_FLOAT
    // C double   --> MPI_DOUBLE
    // C uint32_t --> MPI_UINT32_T
    
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(shift, len_s, MPI_FLOAT, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(proj,  len_p, MPI_FLOAT, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
}

// -----------------------------------------------------------------------------
void output_buckets(                // output buckets to disk
    int   rank,                         // mpi rank
    int   n,                            // number of buckets
    const int  *bktset,                 // bucket set
    const u64  *bktpos,                 // bucket position
    const char *prefix)                 // prefix path
{
    char fname[100]; sprintf(fname, "%s_buckets_%d", prefix, rank);
    FILE *fp = fopen(fname, "wb");
    if (!fp) { printf("Could not open %s\n", fname); exit(1); }

    fwrite(&n,     sizeof(int), 1,         fp);
    fwrite(bktpos, sizeof(u64), n+1,       fp);
    fwrite(bktset, sizeof(int), bktpos[n], fp);
    fclose(fp);
}

// -----------------------------------------------------------------------------
void output_bins(                   // output bins to disk
    int   rank,                         // mpi rank
    int   n,                            // number of bins
    const int  *binset,                 // bin set
    const u64  *binpos,                 // bin position
    const char *prefix)                 // prefix path
{
    if (rank > 0) return;
    
    char fname[100]; sprintf(fname, "%s_bins_%d", prefix, rank);
    FILE *fp = fopen(fname, "wb");
    if (!fp) { printf("Could not open %s\n", fname); exit(1); }

    fwrite(&n,     sizeof(int), 1,         fp);
    fwrite(binpos, sizeof(u64), n+1,       fp);
    fwrite(binset, sizeof(int), binpos[n], fp);
    fclose(fp);
}

// -----------------------------------------------------------------------------
void output_centroids(              // output centroids to disk
    int   rank,                         // mpi rank
    int   len,                          // len = k * d
    const float *centroids,             // centroids
    const char  *prefix)                // prefix
{
    if (rank > 0) return;

    char fname[100]; sprintf(fname, "%s_centroids_%d.bin", prefix, rank);
    FILE *fp = fopen(fname, "wb");
    if (!fp) { printf("Could not open %s\n", fname); exit(1); }
    
    fwrite(centroids, sizeof(float), len, fp);
    fclose(fp);
}

} // end namespace clustering
