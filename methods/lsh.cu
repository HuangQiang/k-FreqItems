#include "hip/hip_runtime.h"
#include "lsh.cuh"

namespace clustering {

// -----------------------------------------------------------------------------
int uni_hash[100] = { 7,73,97,751,397,251,769,431,139,167,997,367,421,569,821,
    733,163,947,739,941,751,41,131,293,709,233,181,223,263,929,151,541,733,89,
    907,373,101,281,149,859,647,911,59,653,257,907,947,197,953,647,113,19,233,
    313,599,929,991,743,421,907,317,571,383,401,797,971,719,397,739,787,239,
    439,967,149,587,73,971,23,31,223,419,431,379,167,479,677,269,257,571,67,
    317,947,79,271,919,929,251,173,617,29, };

// -----------------------------------------------------------------------------
int fixedcoeffA[1000] = { 6517155,4843265,456393,696809,3033337,737017,4568968,
    6240157,6816902,166900,8267017,8054957,3324705,3747952,8377086,2952217,
    3672878,7327488,2742231,2355981,3664319,4042611,2957148,5139656,4396424,
    7911966,2933883,7713940,6489839,4149293,1517455,3006993,8992559,1973849,
    3703803,2025895,2710866,8272771,8266052,9527768,8439671,6533068,7582724,
    1764374,281018,5959809,4716592,3953896,3287295,7458823,6309878,6951614,
    1501433,9267027,2091268,5897857,7178991,5025152,3611796,3668829,9174445,
    5129252,6675823,8167002,7103101,379624,192895,9813967,8652395,8458948,
    9341734,7092065,4992014,6924457,8856439,5273033,2884264,3573029,9226930,
    6171559,1031851,5536806,3123172,2533284,4803832,5214440,8431142,1982822,
    239590,2042936,5651651,9414036,7172189,2327472,7581036,4275288,2707096,
    7773932,4089254,1359490,6232879,3430987,8451555,1224891,355442,7307993,
    6497924,3239706,881021,5724852,9411266,1912872,1261657,2534436,4446157,
    6065489,7748877,2877297,8048312,7988467,4920234,3699961,7402501,2092421,
    6027434,4983536,6367709,8734531,2757467,456962,94019,8990346,3887949,
    8545575,215235,4243391,5853566,6713160,7483098,6734587,2438011,6894362,
    8647460,3699668,9428798,3093615,9765158,7177673,5970912,7813468,5166139,
    891145,1513428,2568639,2983566,7540862,7552176,9351276,6275392,309641,
    9808238,6369412,9299988,3696185,4914985,9515223,7939577,768550,6228382,
    5422673,7503137,8666393,2317033,6150596,2366060,1745830,9244211,2131216,
    8923504,5215122,9944685,4089641,6106267,1458111,6658280,9089833,8998974,
    4210455,8441107,5274364,4520096,8249344,1643775,3820082,1945528,6558760,
    3335304,9885105,7327311,9563687,5307776,4830447,8230078,7624810,981041,
    596137,9370640,225250,2727353,8294142,5440372,2672036,2383782,1546637,
    4130148,9042063,636469,3129120,3252516,9077577,8403485,7772613,7326919,
    47258,1592694,9272447,6606019,4927999,9157551,3933328,4491684,4465325,
    8763776,2721761,2090134,9744817,3317898,1460773,9970068,6045252,9754915,
    5410438,8717289,2138696,6957076,2847435,1180757,7593546,5976556,4433274,
    6671121,4380039,2205886,3998039,4427298,3798581,3270484,1033316,8726580,
    2428033,4966645,3218263,6893359,3730419,5940025,8983493,3475235,9257924,
    444264,3445301,5303175,199178,8855740,4020462,2337875,5812814,6867897,
    3518633,3406358,2844452,7951907,77478,7224491,157792,4075517,1651788,
    3956373,7346002,2685104,2682952,9774036,7651750,5901215,6667393,1382167,
    1841239,5650885,4857402,1099162,6095150,6621590,5417718,5397056,4130962,
    4297141,2733953,2387360,8773507,4167849,1718983,1533126,5517738,9351848,
    6482261,6071894,3060709,5361062,9389566,8911599,340876,7131024,1151684,
    7542142,334077,4599347,4280534,3309643,2333109,6801555,5030420,3289488,
    5566932,6569555,9408667,8477025,8689893,2824371,5126431,9766766,2076503,
    7513567,498585,6155307,3484700,1647063,5830310,6606247,3587099,6353738,
    8656955,2745308,438810,7188423,5936191,1025747,1321302,4369730,9362522,
    4651580,3481898,1362340,1235192,709447,1144388,7818056,1893947,7148188,
    6133041,7897996,1291800,81924,7463055,4432987,4976690,9672797,5383208,
    7770794,9003723,1440690,5543425,9949452,9822109,9770126,1434250,8024381,
    6591864,5339149,2557958,880495,8856701,5993234,1588428,858308,2244267,
    740260,1557744,5225061,7787350,7335001,2663309,1947648,3465686,3310091,
    7058932,3558290,4743876,155541,3315550,6924454,3312847,3347507,8354425,
    6250441,3567755,5363374,8944824,5486235,4597818,8434152,2986909,7572382,
    1543964,4063088,8741772,419330,8652255,6925208,2339404,3148453,6761091,
    2138221,5822848,8116422,6420780,2077415,8543014,7138474,7377720,4946492,
    2189103,8174109,9132657,7848036,8274965,7379219,9508314,741969,7141895,
    5751826,5408964,2970363,7479037,4038955,1310397,976246,4351267,4340856,
    4249672,4955985,3185379,7560692,1852643,8024853,7161051,2557212,1605919,
    6521991,5636411,3941710,9546392,446307,4068310,7381581,6976053,842617,
    4672768,2855124,6705968,5519949,4656655,6437032,3604735,5347527,3519968,
    6187953,4207191,6964017,1421421,6593482,6683163,8558905,5839407,7881342,
    2294005,6412393,1478450,3213231,5808448,4324330,7849994,2394775,7146540,
    332685,3704389,6965433,4362265,1743482,6938365,2747047,3026113,6183028,
    2008864,9587747,7957016,7145726,4424359,7864194,8899292,138859,5123045,
    2991232,2628628,9686982,4359536,8799362,1593467,1890211,4378647,1419641,
    5390034,8142671,8105867,1385291,4203743,9608838,8628440,8812097,7334782,
    824060,933105,9079967,4605809,6128900,2448135,1320117,6113907,7652561,
    5469653,1224931,561999,9000182,3433837,447505,6311355,2462937,894985,
    9771030,2513080,4485248,3448470,2799382,5268153,3782149,9489793,411199,
    5840848,9242230,7223658,43303,961299,779471,4049142,7357077,6450430,
    1523435,7647690,9461276,4774759,3653647,8250503,4316343,3476226,3366034,
    3248307,4234895,5915576,6065889,5564865,6526938,5265290,1914194,2373188,
    5828821,4618893,1412498,3547312,6960746,9287412,7723420,3736116,6745323,
    6972679,9616407,6234176,4552838,4193716,4412444,7842500,3714850,9723731,
    5330788,856187,294580,2618312,2679315,9976314,6006117,2235849,9486536,
    1765868,9969490,506405,2526487,4623198,810297,1368683,7617971,8432617,
    2368467,5930550,7405356,4472419,2855854,9033996,5178293,6381763,6759446,
    9529715,5237072,5527268,8499840,2188453,9120819,4030360,1671999,6537067,
    5867771,2488826,7717753,1066014,2322684,8043827,4335943,8368061,1726454,
    5620384,791278,6344103,8429907,1401382,335371,9179259,8947933,4926766,
    3190905,9561731,3440555,4595718,5179098,7349513,6859106,7131845,778733,
    7562480,3850410,9310543,6902310,388922,157885,2515928,1308466,8862379,
    9683215,2525519,6421874,1767864,2844927,931363,2375655,2832297,7457705,
    411844,632641,210255,6116452,4139315,4803841,2173937,6459789,9349367,
    4574486,6162643,16938,1043071,5587848,8618656,6018207,7405566,9831264,
    7441372,342003,3372631,4582379,9632898,246023,3923831,99495,2239921,
    7180478,2509024,7158359,5813878,158178,9827412,835263,5937155,8849469,
    6323109,4856354,4474015,4938270,9922885,8484318,5154195,5445797,8433121,
    8790898,5780720,7466441,8938516,2876914,9478969,4185457,5158230,1758082,
    4144187,8030089,1734115,6229877,2179201,4551528,4279829,4859319,2291500,
    5668681,627462,6105687,271977,5908507,9332953,5062450,4362939,7544210,
    9606945,2985363,5449761,2398466,879315,9790138,9583543,4400831,4004412,
    317434,797659,9706594,3892528,3607749,7861582,4352520,4926045,9254672,
    4874772,105394,8910921,8924901,5354350,8705468,3832248,3635150,5248841,
    4760517,8110786,7396835,9520224,3440894,9087236,4822074,4348702,9716093,
    4823000,1754086,1966725,9536111,9482454,428877,6513003,3872663,3557659,
    596157,3018851,7524150,6732615,6068093,7273691,4340939,4425178,2274039,
    6554576,2190874,4981409,768323,7809143,6118201,5910482,4444683,751407,
    1035419,4315056,8565338,2887695,3685436,9001300,3704775,7769623,1592904,
    8515399,4245443,7002212,7691343,7077999,1842026,4104857,5313428,9980955,
    8659,6785042,181523,9653567,1342217,9781946,174221,5296348,4621885,
    4783305,5211253,4449252,9690710,9930722,6974021,8882902,2780326,676772,
    1454285,6848146,7753528,5731622,4927434,2347411,3326527,1862545,8992020,
    5591057,4865904,3686866,5433456,8822603,8237711,1175630,9829714,2488496,
    4653865,5510989,9248085,9130616,4989116,1112391,7304956,2663864,2225227,
    4010783,8920497,5872463,2332760,6850201,5071866,9745714,7558812,9307029,
    3115102,9381508,9430805,2085352,7197346,8906619,1099944,7081253,5750786,
    7178646,1808,246017,3367542,4799404,5248433,7733250,2544076,6467751,
    7608656,8710639,3302316,4228573,534658,4422699,4263542,4368988,5630191,
    9339022,4606735,9924087,8244856,8218700,4772094,2763811,3239552,7186752,
    9091706,944127,2725151,5911806,6968171,1254040,4895798,1596094,4731486,
    9357698,9177516,3765272,2262849,6330360,8003531,2044268,4932435,6721599,
    8355220,4646276,2914198,7921751,6493377,7312528,1038024,4201202,4246623,
    4165149,4495555,2011871,4012619,9594321,7044532,6602638,8949325,723142,
    7149229,9673518,8162736,4867467,3656270,9416299,7035143,4412364,4076956,
    4029783,480372,7034628,9940023,2924655,4886179,752869,2292688,5678013,
    7412430,9656184,9614148,8971592,2653930,8179987,9808188,4820460,5833872,
    7813463,5492352,4663241,};

// -----------------------------------------------------------------------------
int fixedcoeffB[1000] = { 4912045,1121710,6068228,4062045,6585752,6977557,
    7609116,2860722,7040660,4123844,2006871,6934457,1107623,6983831,9381675,
    3138533,5024471,7871574,2539842,1269105,9140341,9382344,1533366,6759648,
    4408022,531611,511108,7122767,9375412,3176618,1217891,4287455,4298328,
    7286120,8349501,884079,4263675,5958615,3744802,1304334,82457,5751674,
    8238791,1190081,2735503,7620464,4328614,7759974,5492036,6868457,9029079,
    4632376,6250800,562444,1392022,658820,1094056,1903130,7781588,469466,
    5079749,8999479,4756922,9378077,6285597,3106421,262155,549271,9065037,
    4006957,1853606,9147495,9758631,92396,337574,2494132,7712861,4666189,
    254105,3204896,1534644,9283185,7837272,7785444,9845630,9229294,8444265,
    939684,1132423,6225851,1409151,6212172,5225329,6166074,5590248,1510924,
    9272496,5852403,2060196,8337531,9859360,3913802,7485025,9617990,4006199,
    7822599,2112121,1719058,2488786,2366226,4923954,4023431,1649410,2761224,
    1808874,1495038,1990517,253137,2434723,3122940,6478988,3843875,9335113,
    1704315,9947,4925359,3215240,9282444,777761,5275437,7619974,637120,9189240,
    5104997,255108,3195437,2927595,2367229,4914495,5416381,4733455,9838449,
    9439813,6382866,2599672,1248685,7877904,4590189,1501822,312626,7713130,
    7980811,4156501,7048241,9685127,4166449,1973599,2900366,3448891,2751360,
    8175803,1068864,3388481,7365041,6173861,3643589,560476,9101457,6010818,
    5474972,4517837,744272,5313419,3957648,7127138,7913092,5206333,5005041,
    2503280,6708156,5317667,216408,4688966,9474169,7264650,4374092,3640617,
    9238250,7274458,7089509,1989609,5450259,8158373,5378090,2815299,4332233,
    9021679,3375776,3433688,5032495,8850748,7951525,5776768,4164166,1909172,
    2903904,2077256,7115506,7908946,4580536,3823661,3226612,4796945,8512627,
    2700780,2061594,2886717,6341397,1299842,161174,3430904,3289451,5611433,
    1589275,8667541,8426733,5921509,7689218,1802507,9355197,2721712,653253,
    7306721,8498481,4817419,9215893,1402384,6894676,6331397,9311330,1475211,
    155056,2537940,6272156,8667684,5238721,8333750,1554400,1580116,9633592,
    1715574,5011020,2923042,7327008,6600296,1590581,5753739,2521803,9279800,
    7556246,1876999,2001511,8209500,9183720,499991,3026918,8399612,1902375,
    9921594,4731008,1213703,1396803,4886065,3751644,7668960,3553748,8990365,
    6002709,5108148,570480,5636300,6823722,5581500,8559342,4150728,2181795,
    149922,9904467,4703598,9429723,7460712,6580598,1431232,5670210,5764317,
    1931223,8697128,4163927,3833599,8618721,886945,6696083,433174,9582758,
    3610496,7942945,4467314,815695,4674157,2570543,1256584,6200637,7003284,
    2368930,4474118,2693484,8478426,758118,3520528,3208740,3074829,9518383,
    9925190,1709991,2053060,6132473,9798271,2661491,8815370,2873448,8091129,
    8650550,4956498,5481154,9749543,5608048,9841577,366513,5460507,4366113,
    6357037,823209,2831694,2956862,8515119,4936550,1397241,3175802,2995766,
    4830762,7346847,5569678,5565240,6955189,8012694,8539313,1819887,525037,
    2847221,6298642,9312207,7968587,4822291,7063185,9801342,563364,2332756,
    4336611,2069231,8923743,2613101,4451654,6825577,9084566,8458587,1035137,
    6618400,8480240,148110,9258746,7437371,7097639,5955150,7668122,2726049,
    8106305,4054919,9772136,7534647,553292,4239909,3764027,592151,2298998,
    7787691,689711,1379852,4995187,9221600,5359382,9255172,7286797,6023643,
    4190585,5915633,9551524,6332304,1013849,5559817,7664714,4870590,8572568,
    6141775,3585442,6231063,7311022,2410509,8796325,1656750,1992441,7730785,
    7727923,4859514,987542,2054619,3766837,7006679,3175649,5907159,8511826,
    5705914,7624456,420665,6550573,9407689,9649797,86978,1201562,6119491,
    5365821,841357,5099317,1472061,996488,5036056,9517440,5331161,8858391,
    3562675,7829240,3951978,108836,4856756,8985446,3499500,9150355,3483924,
    2188891,2524636,2335204,2693693,8302660,8994538,3776679,2570014,5527575,
    9084955,560946,6946808,4917956,3897658,4787162,8317981,2337565,161432,
    8403309,6525411,879977,3867019,8077161,6262573,2959023,9241146,9289236,
    3066360,6634259,8275501,4345598,8397473,7436099,1355931,1414028,2545375,
    3401290,6950659,2849425,5083185,8560758,9328416,512282,2057185,9358792,
    8567960,6926167,387552,5524337,201672,9624823,9232369,9159439,4504563,
    9396259,7817311,3912077,5807009,9540501,1570899,8196879,2603681,7104291,
    9485310,4734713,9566993,2434028,7385649,4982389,3602330,1477958,2335662,
    8565367,9814393,6006865,7005158,1524149,9326787,2600069,1257067,8499078,
    1694318,9623421,2526571,970177,8984455,7548541,7838169,9462788,4219773,
    9037851,9673395,833768,9190439,7812278,768619,1271565,7244459,1542903,
    5500794,5899930,4961214,5805365,9562377,451943,737805,5007080,51792,
    9411380,9274684,9689735,3535012,2506239,1856284,550501,6562221,1842325,
    9898900,4322817,2915842,3243424,4264551,4064846,4091350,3558572,699883,
    3409337,4209606,5014365,9964717,3422965,2015295,8050372,2912979,1375469,
    7147947,9482467,2869396,5728840,7023538,326779,2535718,1958314,9651691,
    2811212,2537544,6143344,1499466,6467032,6566904,241264,2461393,5127347,
    3271043,2291533,143598,8018000,6354427,9424638,6948992,5622524,8470025,
    4039374,5959583,4381934,8348905,6241336,4346928,6141085,2224869,7912550,
    1050929,5047782,5906863,3411473,6708268,2342390,799863,5147782,6888168,
    9285932,4152913,6047369,6895950,6117810,621070,1489286,9960909,9077354,
    1009486,8723625,1522822,2896045,7447776,9082129,7573542,9573015,3895328,
    226293,7808837,9912097,374463,101905,7938448,6203142,1668084,6665686,
    5992779,5513589,7605557,7776498,2963778,6905552,2969896,7650384,3475947,
    9208639,8963862,4185780,8489801,5850242,5315687,3348790,2077658,4749021,
    3458096,4862313,9875031,3280980,8899708,3623659,8493843,6226064,2142840,
    4360014,800765,790525,2726183,7424917,3001693,4487055,4699456,2881663,
    1117231,7419413,7094006,2392592,3875368,3045553,9030499,6650847,7615821,
    5947715,1580686,5145843,5546568,1042168,8982264,4820509,86473,116432,
    3282992,2768326,1127679,3176565,3219443,9989441,7268783,1177266,9309786,
    3431848,5956006,5102377,8397845,5037764,2162806,5664795,7663219,2604156,
    4246998,8949075,5608132,8301703,4875671,1726383,5802077,9041810,9632245,
    6696351,4067072,6514406,6888310,1675674,612321,3342425,60549,2325759,
    1230016,8545594,106430,291143,5430206,7666588,3477139,3092539,6097053,
    3774087,1786145,7536890,9729590,7257784,3170815,5458818,211005,506517,
    4276388,6225940,130611,4769044,8255467,1151959,7468527,7337882,375805,
    1653137,5754087,9895532,8494464,3325731,9358762,9022901,4629906,3341743,
    9141369,8368266,5972368,1749354,4446269,2695565,3962007,4927468,3259622,
    8640994,177349,459553,5798972,3171260,4731294,5189393,4623789,1269497,
    3077882,4037051,2106086,2346224,1081317,5784185,2096690,6095024,6391903,
    1810149,6037029,2619751,1494412,4640985,1366801,6714357,8929333,1968833,
    7484022,7103511,1588746,2906277,7312873,5959474,5555876,5895649,2751031,
    2361227,6324542,363643,2912694,2136025,7829406,2877445,8419913,3878766,
    2868650,3694094,5786342,3886298,7423190,2935121,8297344,4419663,2298559,
    2313227,1861304,7053278,8554109,5004137,9911677,6982679,5667944,2961849,
    330339,2796501,6630091,7927749,1027969,9549318,7130716,8537779,6311796,
    8682729,5521779,1839924,2569768,8213782,9499375,9691995,8023775,7583241,
    8220117,9581298,7858400,2574612,993161,3981135,6703448,2170859,6350729,
    7658682,2833385,7158207,7048092,2162473,9338696,9725731,5256703,6806897,
    8437644,8104217,493899,6735760,4749215,9422572,2917401,20440,2953716,
    8083545,4028847,5073673,9991730,926354,4106432,8363534,7114758,1974880,
    8553102,4806002,7617257,4402260,1309826,6285443,1601345,4265349,4615005,
    8997733,4138425,9855167,7657337,788931,4686451,2380750,1474449,6541868,
    1415325,9626363,5366826,6988692,280794,2598361,593061,9502532,1520180,
    2976341,7576398,3949610,6529082,5502400,3517111,7095236,2988394,7657585,
    2633405,5323390,5900316,7809996,9517845,1224862,6670226,215540,7966474,
    6509471,495509,2499086,4056138,8945054,6315920,5399752,1248914,1863522,
    8644892,3642606,6986045,9200760,9068669,7902893,3581448,1530697,6424234,
    8089575,3624103,7524904,3637265,8889347,3647797,2035408,426386,455424,
    545480,1068700,9072483,6649290,3383955,6730886,2209769,288560,1855088,
    5773895,738118,378531,};

// -----------------------------------------------------------------------------
template<class DType>
__global__ void sign_minhash(       // calc minhash value
    int   batch,                        // number of data points/buckets
    int   sign_size,                    // sign_size = l * k
    int   prime,                        // prime number for dimension
    const int   *d_coeA,                // permutation hash coefficients
    const int   *d_coeB,                // permutation hash coefficients
    const DType *d_set,                 // data/bucket set
    const u64   *d_pos,                 // data/bucket position
    int   *d_sig)                       // hash values of data/bucket (return)
{
    // d_sig has <batch> chunks, and each chunk has <sign_size> hash values
    u64 tid = (u64) blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < (u64) batch * sign_size) {
        int   did = tid / sign_size;            // get data/bucket id
        int   num = get_length(did, d_pos);     // get data/bucket size
        const DType *data = d_set + d_pos[did]; // get data/bucket
        
        int sid  = tid % sign_size; // signature id
        int coeA = d_coeA[sid], coeB = d_coeB[sid];
        
        // scan and find the minimum order as hash value of this data/bucket
        int val = ((u64) data[0] * coeA + coeB) % prime;
        for (int i = 1; i < num; ++i) {
            int tmp = ((u64) data[i] * coeA + coeB) % prime;
            if (tmp < val) val = tmp;
        }
        d_sig[tid] = val;
    }
}

// -----------------------------------------------------------------------------
__global__ void sign_concat(        // universal hash to get k-concatenation
    int   batch,                        // number of batch data
    int   l,                            // number of hash values
    int   k,                            // k-concatenation
    int   prime,                        // prime number for data/buckets
    const int *d_sig,                   // signatures
    const int *d_hash,                  // universal hash function
    int   *d_val)                       // hash values (return)
{
    // d_val has <batch> chunks, and each chunk has <l> signatures
    u64 tid = (u64) blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < (u64) batch * l) {
        const int *sig = d_sig + (u64) tid*k; // get signatures
        
        // universal hashing for k-concatenation
        u64 val = 0UL;
        for (int i = 0; i < k; ++i) {
            // val & MAX_UINT32: low-32-bit, val >> 32: high-32-bit
            val += (u64) d_hash[i] * (u64) sig[i];
            val = (val & MAX_UINT32) + 5 * (val >> 32);
            if (val > UINT32_PRIME) val -= UINT32_PRIME;
        };
        d_val[tid] = val % prime;
    }
}

// -----------------------------------------------------------------------------
__global__ void combine(            // combine two k-concatenations together
    int   batch,                        // number of batch data
    int   m,                            // number of hash tables
    int   l,                            // number of hash values
    int   prime,                        // prime number
    const int *d_val,                   // hash values
    int   *d_res)                       // hash results (return)
{
    // d_res has <m> chunks, and each chunk has <batch> signatures
    u64 tid = (u64) blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < (u64) batch) {
        const int *val = d_val + (u64) tid*l; // get hash values
        
        int cnt = 0;
        for (int i = 0; i < l; ++i) {
            for (int j = i+1; j < l; ++j) {
                u64 pos = (u64) cnt * batch + tid; // get position
                d_res[pos] = ((u64) val[i] + val[j]) % prime;
                if (++cnt >= m) break;
            }
            if (cnt >= m) break;
        }
    }
}

// -----------------------------------------------------------------------------
template<class DType>
void minhash(                       // minwise hashing (pair-wise)
    int   rank,                         // MPI rank
    int   n,                            // number of data points/buckets
    int   n_prime,                      // prime number for data/buckets
    int   d_prime,                      // prime number for dimension
    int   m,                            // #hash tables
    int   h,                            // #concatenated hash func
    const DType *dataset,               // data/bucket set
    const u64   *datapos,               // data/bucket position
    int   *hash_results)                // hash results (return)
{
    hipSetDevice(DEVICE_LOCAL_RANK);
    
    assert(h % 2 == 0);
    int k = h / 2;
    int l = (int) ceil(sqrt(2.0f*m)) + 1;
    // only have 1,000 static coeff values and 100 uni_hash values
    if (l*k > MAX_NUM_HASH || k > 100) exit(1); 
    
    // init hash parameters 
    int *d_coeA; hipMalloc((void**)&d_coeA, sizeof(int)*l*k);
    int *d_coeB; hipMalloc((void**)&d_coeB, sizeof(int)*l*k);
    int *d_hash; hipMalloc((void**)&d_hash, sizeof(int)*k);
    
    hipMemcpy(d_coeA, fixedcoeffA, sizeof(int)*l*k, hipMemcpyHostToDevice);
    hipMemcpy(d_coeB, fixedcoeffB, sizeof(int)*l*k, hipMemcpyHostToDevice);
    hipMemcpy(d_hash, uni_hash,    sizeof(int)*k,   hipMemcpyHostToDevice);
    
    // mem_avail = totoal_size - memory(d_coeA + d_coeB + d_hash)
    u64 mem_avail = GPU_MEMORY_LIMIT - sizeof(int)*(l*k*2+k);
    u64 mem_usage = 0UL, n_set = 0UL;
    int n_pos = 0, batch = 0, start = 0;
    
    for (int i = 0; i <= n; ++i) {
        // ---------------------------------------------------------------------
        //  calculate memory usage requirement if adding one more data
        // ---------------------------------------------------------------------
        if (i < n) {
            // d_set + d_pos + d_sig + d_val + d_res
            n_set = datapos[i+1] - datapos[start];
            n_pos = batch + 2;
            mem_usage = sizeof(DType)*n_set + sizeof(u64)*n_pos + 
                sizeof(int)*((u64) l*k+l+m)*(batch+1);
        }
        // ---------------------------------------------------------------------
        //  parallel minhash for bucket if over mem_avail or end
        // ---------------------------------------------------------------------
        if (mem_usage > mem_avail || i == n) {
            // d_set + d_pos + d_sig + d_val + d_res
            n_set = datapos[i] - datapos[start];
            n_pos = batch + 1;
            mem_usage = sizeof(DType)*n_set + sizeof(u64)*n_pos + 
                sizeof(int)*((u64) l*k+l+m)*batch;
#ifdef DEBUG_INFO
            printf("Rank #%d: n=%d, i=%d, batch=%d, mem_usage=%lu, mem_avail=%lu\n", 
                rank, n, i, batch, mem_usage, mem_avail);
#endif
            // cuda allocation and memory copy from CPU to GPU
            const DType *h_set = dataset + datapos[start]; // dataset at host
            int *h_res = new int[(u64) batch*m];           // results at host
            u64 *h_pos = new u64[n_pos];              // data postion at host
            copy_pos(n_pos, datapos + start, h_pos);

            // cuda allocation: dataset, data position, signatures, hash results
            DType *d_set; hipMalloc((void**) &d_set, sizeof(DType)*n_set);
            u64   *d_pos; hipMalloc((void**) &d_pos, sizeof(u64)*n_pos);
            int   *d_sig; hipMalloc((void**) &d_sig, sizeof(int)*batch*l*k);
            int   *d_val; hipMalloc((void**) &d_val, sizeof(int)*batch*l);
            int   *d_res; hipMalloc((void**) &d_res, sizeof(int)*batch*m);
            
            hipMemcpy(d_set, h_set, sizeof(DType)*n_set, hipMemcpyHostToDevice);
            hipMemcpy(d_pos, h_pos, sizeof(u64)*n_pos,   hipMemcpyHostToDevice);
            
            // calc minhash values for batch data by GPUs
            int block = BLOCK_SIZE;
            int grid  = ((u64) batch*l*k + block-1) / block;
            sign_minhash<DType><<<grid, block>>>(batch, l*k, d_prime, d_coeA, 
                d_coeB, d_set, d_pos, d_sig);
            
            // calc the k-concatenation by GPUs
            grid = ((u64) batch*l + block-1) / block;
            sign_concat<<<grid, block>>>(batch, l, k, n_prime, d_sig, d_hash, d_val);
            
            // calc the final m hash values by GPUs
            grid = ((u64) batch + block-1) / block;
            combine<<<grid, block>>>(batch, m, l, n_prime, d_val, d_res);
            
            hipMemcpy(h_res, d_res, sizeof(int)*batch*m, hipMemcpyDeviceToHost);
            // hipDeviceSynchronize();
            
            for (int j = 0; j < m; ++j) {
                const int *res = h_res + (u64) j*batch;
                int *hash_res  = hash_results + (u64) j*n+start;
                std::copy(res, res + batch, hash_res);
            }
            // release space
            hipFree(d_set); hipFree(d_pos);
            hipFree(d_res); hipFree(d_val); hipFree(d_sig);
            delete[] h_pos;  delete[] h_res;
            
            // update local parameters for next minhash
            start += batch; batch = 0;
        }
        if (i < n) ++batch;
    }
    // release space
    assert(start == n);
    hipFree(d_hash); hipFree(d_coeA); hipFree(d_coeB);
}

// -----------------------------------------------------------------------------
template void minhash(              // minwise hashing (pair-wise)
    int   rank,                         // MPI rank
    int   n,                            // number of data points/buckets
    int   n_prime,                      // prime number for data/buckets
    int   d_prime,                      // prime number for dimension
    int   m,                            // #hash tables
    int   h,                            // #concatenated hash func
    const u08 *dataset,                 // data/bucket set
    const u64 *datapos,                 // data/bucket position
    int   *hash_results);               // hash results (return)
    
// -----------------------------------------------------------------------------
template void minhash(              // minwise hashing (pair-wise)
    int   rank,                         // MPI rank
    int   n,                            // number of data points/buckets
    int   n_prime,                      // prime number for data/buckets
    int   d_prime,                      // prime number for dimension
    int   m,                            // #hash tables
    int   h,                            // #concatenated hash func
    const u16 *dataset,                 // data/bucket set
    const u64 *datapos,                 // data/bucket position
    int   *hash_results);               // hash results (return)
    
// -----------------------------------------------------------------------------
template void minhash(              // minwise hashing (pair-wise)
    int   rank,                         // MPI rank
    int   n,                            // number of data points/buckets
    int   n_prime,                      // prime number for data/buckets
    int   d_prime,                      // prime number for dimension
    int   m,                            // #hash tables
    int   h,                            // #concatenated hash func
    const int *dataset,                 // data/bucket set
    const u64 *datapos,                 // data/bucket position
    int   *hash_results);               // hash results (return)
    
// -----------------------------------------------------------------------------
template void minhash(              // minwise hashing (pair-wise)
    int   rank,                         // MPI rank
    int   n,                            // number of data points/buckets
    int   n_prime,                      // prime number for data/buckets
    int   d_prime,                      // prime number for dimension
    int   m,                            // #hash tables
    int   h,                            // #concatenated hash func
    const f32 *dataset,                 // data/bucket set
    const u64 *datapos,                 // data/bucket position
    int   *hash_results);               // hash results (return)

// -----------------------------------------------------------------------------
template<class DType>
__global__ void signature(          // calc signature
    int   batch,                        // number of batch data
    int   sign_size,                    // sign_size = l*k,
    int   d,                            // data dimension
    float w,                            // bucket width
    const DType *d_data,                // batch data
    const float *d_proj,                // random projection
    const float *d_shift,               // random shift
    int   *d_sig)                       // signatures (return)
{
    // d_sig has <batch> chunks, and each chunk has <sign_size> signatures
    u64 tid = (u64) blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < (u64) batch * sign_size) {
        int   did = tid / sign_size; // get data ID
        int   pid = tid % sign_size; // get proj ID
        
        const DType *data = d_data + (u64) did*d;
        const float *proj = d_proj + pid*d;
        
        float shift = d_shift[pid];
        float val = thrust::inner_product(thrust::device, data, data+d, proj, 0.0f);
        d_sig[tid] = (int) floor((val + shift) / w);
    }
}

// -----------------------------------------------------------------------------
__global__ void concatenation(      // universal hash to get k-concatenation
    int   batch,                        // number of batch data
    int   l,                            // number of hash values
    int   k,                            // k-concatenation
    int   prime,                        // prime number
    const int *d_sig,                   // signatures
    const int *d_hash,                  // universal hash function
    int   *d_val)                       // hash values (return)
{
    // d_val has <batch> chunks, and each chunk has <l> signatures
    u64 tid = (u64) blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < (u64) batch * l) {
        const int *sig = d_sig + (u64) tid*k; // get signature
        
        // universal hashing for k-concatenation
        u64 val = 0UL;
        for (int i = 0; i < k; ++i) {
            // val & MAX_UINT32: low-32-bit, val >> 32: high-32-bit
            // NOTE: sig[i] can be negative !!!
            val += (u64) d_hash[i] * (u32) sig[i];
            val = (val & MAX_UINT32) + 5 * (val >> 32);
            if (val > UINT32_PRIME) val -= UINT32_PRIME;
        }
        d_val[tid] = val % prime;
    }
}

// -----------------------------------------------------------------------------
template<class DType>
void e2lsh(                         // calc hash results using e2lsh
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   d,                            // data dimension
    int   prime,                        // prime number
    int   m,                            // number of hash tables
    int   h,                            // number of concat hash functions
    float w,                            // bucket width
    const float *proj,                  // random projection, m*h*d
    const float *shift,                 // random shift, m*h
    const DType *dataset,               // data set
    int   *hash_results)                // hash results (return)
{
    hipSetDevice(DEVICE_LOCAL_RANK);
    
    assert(h % 2 == 0);
    int k = h / 2;
    int l = (int) ceil(sqrt(2.0f*m)) + 1;
    // only have 1,000 static coeff values and 100 uni_hash values
    if (l*k > MAX_NUM_HASH || k > 100) exit(1);
    
    // init hash parameters
    float *d_proj;  hipMalloc((void**)&d_proj,  sizeof(float)*l*k*d);
    float *d_shift; hipMalloc((void**)&d_shift, sizeof(float)*l*k);
    int   *d_hash;  hipMalloc((void**)&d_hash,  sizeof(int)*k);
    
    hipMemcpy(d_proj,  proj, sizeof(float)*l*k*d, hipMemcpyHostToDevice);
    hipMemcpy(d_shift, shift,  sizeof(float)*l*k, hipMemcpyHostToDevice);
    hipMemcpy(d_hash,  uni_hash,   sizeof(int)*k, hipMemcpyHostToDevice);
    
    // mem_avail = total_size - memory(d_proj + d_shift + d_uni_hash)
    // batch = mem_avail / memory(d_data + d_sig + d_val + d_res)
    u64 mem_avail = GPU_MEMORY_LIMIT - (sizeof(float)*l*k*(d+1) + sizeof(int)*k);
    u64 mem_one_cost = sizeof(DType)*d + sizeof(int)*(l*k+l+m);
    u64 batch = mem_avail / mem_one_cost;
    if (batch > n) batch = n;
    
    // init parameters for batch data
    int   *h_res = new int[(u64) batch*m];
    DType *d_data; hipMalloc((void**) &d_data, sizeof(DType)*batch*d);
    int   *d_sig;  hipMalloc((void**) &d_sig,  sizeof(int)*batch*l*k);
    int   *d_val;  hipMalloc((void**) &d_val,  sizeof(int)*batch*l);
    int   *d_res;  hipMalloc((void**) &d_res,  sizeof(int)*batch*m);

    // compute random projections
    for (int i = 0; i < n; i += batch) {
        if (i+batch > n) batch = n-i;
#ifdef DEBUG_INFO
        printf("Rank #%d: n=%d, i=%d, batch=%d\n", rank, n, i, batch);
#endif
        // copy data from CPU memory to GPU memory
        const DType *h_data = dataset + (u64) i*d;
        hipMemcpy(d_data, h_data, sizeof(DType)*batch*d, hipMemcpyHostToDevice);
        
        // calc h_{a,b}(o)=floor((<a,o>+b)/w) by GPUs
        int block = BLOCK_SIZE;
        int grid  = ((u64) batch*l*k + block-1) / block;
        signature<DType><<<grid, block>>>(batch, l*k, d, w, d_data, d_proj,
            d_shift, d_sig);
        
        // calc the k-concatenation by GPUs
        grid = ((u64) batch*l + block-1) / block;
        concatenation<<<grid, block>>>(batch, l, k, prime, d_sig, d_hash, d_val);
        
        // calc the final m hash values by GPUs
        grid = ((u64) batch + block-1) / block;
        combine<<<grid, block>>>(batch, m, l, prime, d_val, d_res);
        
        // update results
        hipMemcpy(h_res, d_res, sizeof(int)*batch*m, hipMemcpyDeviceToHost);
        // hipDeviceSynchronize();
        
        for (int j = 0; j < m; ++j) {
            const int *res = h_res + (u64) j*batch;
            int *hash_res  = hash_results + (u64) j*n+i;
            std::copy(res, res + batch, hash_res);
        }
    }
    // free memory
    hipFree(d_proj); hipFree(d_shift); hipFree(d_hash);
    hipFree(d_data); hipFree(d_sig); hipFree(d_val); hipFree(d_res);
    delete[] h_res;
}

// -----------------------------------------------------------------------------
template void e2lsh(                // calc hash results using e2lsh
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   d,                            // data dimension
    int   prime,                        // prime number
    int   m,                            // number of hash tables
    int   h,                            // number of concat hash functions
    float w,                            // bucket width
    const float *proj,                  // random projection, m*h*d
    const float *shift,                 // random shift, m*h
    const u08   *dataset,               // data set
    int   *hash_results);               // hash results (return)

// -----------------------------------------------------------------------------
template void e2lsh(                // calc hash results using e2lsh
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   d,                            // data dimension
    int   prime,                        // prime number
    int   m,                            // number of hash tables
    int   h,                            // number of concat hash functions
    float w,                            // bucket width
    const float *proj,                  // random projection, m*h*d
    const float *shift,                 // random shift, m*h
    const u16   *dataset,               // data set
    int   *hash_results);               // hash results (return)

// -----------------------------------------------------------------------------
template void e2lsh(                // calc hash results using e2lsh
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   d,                            // data dimension
    int   prime,                        // prime number
    int   m,                            // number of hash tables
    int   h,                            // number of concat hash functions
    float w,                            // bucket width
    const float *proj,                  // random projection, m*h*d
    const float *shift,                 // random shift, m*h
    const int   *dataset,               // data set
    int   *hash_results);               // hash results (return)

// -----------------------------------------------------------------------------
template void e2lsh(                // calc hash results using e2lsh
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   d,                            // data dimension
    int   prime,                        // prime number
    int   m,                            // number of hash tables
    int   h,                            // number of concat hash functions
    float w,                            // bucket width
    const float *proj,                  // random projection, m*h*d
    const float *shift,                 // random shift, m*h
    const float *dataset,               // data set
    int   *hash_results);               // hash results (return)

} // end namespace clustering
