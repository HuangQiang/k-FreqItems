#include "hip/hip_runtime.h"
#include "bin.h"

namespace clustering {

// -----------------------------------------------------------------------------
int signature_to_index(             // convert signature into a set of bins
    int n,                              // length of signature
    int *signature,                     // signature (allow modify)
    int *index,                         // index of signature (return)
    int *index_key,                     // index key (return)
    int *index_pos)                     // index position (return)
{
    // sort signature and its corresponding index
    thrust::sequence(index, index+n); // init index with 0,1,2,3,...
    thrust::sort_by_key(signature, signature+n, index);
    
    // update index_pos
    int j = 0; index_pos[0] = 0;
    for (int i = 1; i < n; ++i) {
        if (signature[i] != signature[i-1]) {
            index_key[j++] = signature[i-1]; 
            index_pos[j] = i;
        }
    }
    index_key[j++] = signature[n-1];
    index_pos[j] = n;
    return j;
}

// -----------------------------------------------------------------------------
void get_all_point_ids(             // get all point IDs from bin
    int   num,                          // number of bucket IDs in a bin
    const int *bin,                     // bin
    const int *bktset,                  // bucket set
    const u64 *bktpos,                  // bucket position
    int   *tmp_arr,                     // tmp array (allow modify)
    u64   &total_num)                   // total num (return)
{
    // get total_num of point IDs for the buckets in this bin
    total_num = 0UL;
    for (int i = 0; i < num; ++i) {
        int id = bin[i]; total_num += bktpos[id+1] - bktpos[id];
    }
    if (total_num > STATIC_ARRAY_SIZE) return;
    
    // init an array to store all point IDs in this bin
    u64 cnt = 0UL;
    for (int i = 0; i < num; ++i) {
        const u64 *pos = bktpos + bin[i];
        const int *bkt = &bktset[pos[0]];  // get bucket
        int len = (int) (pos[1] - pos[0]); // get bucket len
        
        // copy this bucket (point IDs) to the array
        std::copy(bkt, bkt+len, tmp_arr+cnt);
        cnt += len;
    }
    assert(cnt == total_num);
}

// -----------------------------------------------------------------------------
void get_cand_bin(                  // get cand bin
    int   n,                            // number of distinct pIDs
    int   max_freq,                     // maximum frequency
    int   cand_bin_size,                // total cand_bin size
    float bin_freq,                     // bin freq to get frequent data
    const int *pid,                     // distinct point IDs
    const int *freq,                    // freq of distinct point IDs
    int   *cand_bin,                    // cand_bin (return)
    int   &cand_len)                    // actual length of cand_bin (return)
{
    assert(n <= cand_bin_size);
    int threshold = (int) ceil((double) max_freq*bin_freq);
    
    // get cand_bin & update cand_len
    if (max_freq == 1) {
        cand_len = n;
        std::copy(pid, pid + n, cand_bin);
    } 
    else {
        cand_len = 0;
        for (int i = 0; i < n; ++i) {
            if (freq[i] >= threshold) cand_bin[cand_len++] = pid[i];
        }
    }
    
    // drop this cand_bin if too large
    if (cand_len > CAND_LEN_SIZE) {
        // printf("cand_len=%d, max_freq=%d, thresold=%d\n", cand_len, max_freq, 
        //     threshold);
        cand_len = 0;
    }
}

// -----------------------------------------------------------------------------
int frequent_data(                  // get frequent data from buckets
    int   num,                          // number of bucket IDs in a bin
    int   cand_bin_size,                // total cand_bin size
    float bin_freq,                     // bin freq to get frequent data
    const int *bin,                     // bin
    const int *bktset,                  // bucket set
    const u64 *bktpos,                  // bucket position
    int   *tmp_arr,                     // tmp array (allow modify)
    int   *tmp_pid,                     // tmp pid list (allow modify)
    int   *tmp_freq,                    // tmp freq list (allow modify)
    int   *cand_bin,                    // cand bin (return)
    int   &cand_len)                    // actual length of cand_bin (return)
{
    // get all point IDs from the input bin
    u64 total_num = 0UL;
    get_all_point_ids(num, bin, bktset, bktpos, tmp_arr, total_num);
    
    if (total_num > STATIC_ARRAY_SIZE) {
        // printf("total_num = %lu\n", total_num);
        cand_len = 0;
        return 0;
    }
    
    // get the distinct point IDs and their frequencies
    int n = 0;
    int max_freq = distinct_coord_and_freq<int>(total_num, tmp_arr, tmp_pid, 
        tmp_freq, n);
    
    // get the high frequent point IDs & add them to cand_bin
    get_cand_bin(n, max_freq, cand_bin_size, bin_freq, tmp_pid, tmp_freq, 
        cand_bin, cand_len);
    
    return max_freq;
}

// -----------------------------------------------------------------------------
void add_cand_bin(                  // add cand_bin to bins
    int  len,                           // length of cand_bin
    int  delta,                         // delta
    bool filter,                        // whether filter the points in bin
    int  *cand_bin,                     // cand_bin (allow modify)
    std::vector<int> &binset,           // bin set (return)
    std::vector<u64> &binpos)           // bin position (return)
{
    if (len > delta) {
        // only keep at most MAX_BIN_SIZE point IDs if filter is true
        if (filter && len > MAX_BIN_SIZE) {
            std::random_shuffle(cand_bin, cand_bin+len);
            binpos.push_back(MAX_BIN_SIZE);
            binset.insert(binset.end(), cand_bin, cand_bin+MAX_BIN_SIZE);
        } else {
            binpos.push_back(len);
            binset.insert(binset.end(), cand_bin, cand_bin+len);
        }
    }
}

// -----------------------------------------------------------------------------
void index_to_bins(                 // convert index into bins
    int   num_keys,                     // number of keys
    int   cand_bin_size,                // total cand_bin size
    int   b,                            // threshold of #buckets in a bin
    int   delta,                        // threshold of #points  in a bin
    bool  filter,                       // whether filter the points in bin
    float gbeta,                        // global beta
    const int *bktset,                  // bucket set
    const u64 *bktpos,                  // bucket position
    const int *index,                   // index 
    const int *index_pos,               // index position
    int   *tmp_arr,                     // tmp array (allow modify)
    int   *tmp_pid,                     // tmp pid list (allow modify)
    int   *tmp_freq,                    // tmp freq list (allow modify)
    int   *cand_bin,                    // candidate bin (allow modify)
    std::vector<int> &binset,           // bin set (return)
    std::vector<u64> &binpos)           // bin position (return)
{
    // select bins and add to binset
    int len = 0; // actual length of cand_bin
    for (int i = 0; i < num_keys; ++i) {
        const int *bin = index + index_pos[i]; // get bin
        int num = index_pos[i+1]-index_pos[i]; // get bin num
        if (num <= b) continue; // skip if |bin| <= b
        
        // using global bin freq to get cand_bin
        frequent_data(num, cand_bin_size, gbeta, bin, bktset, bktpos,
            tmp_arr, tmp_pid, tmp_freq, cand_bin, len);
        
        // add cand_bin to bins
        add_cand_bin(len, delta, filter, cand_bin, binset, binpos);
    }
}

// -----------------------------------------------------------------------------
int index_to_buffer(                // convert index into buffer
    int   num_keys,                     // number of keys
    int   cand_bin_size,                // total cand_bin size
    int   max_buff,                     // maximum buffer size
    float lbeta,                        // local beta
    const int *bktset,                  // bucket set
    const u64 *bktpos,                  // bucket position
    const int *index,                   // index 
    const int *index_key,               // index key
    const int *index_pos,               // index position
    int   *tmp_arr,                     // tmp array (allow modify)
    int   *tmp_pid,                     // tmp pid list (allow modify)
    int   *tmp_freq,                    // tmp freq list (allow modify)
    int   *cand_bin,                    // candidate bin (allow modify)
    std::vector<std::vector<int> > &buffer) // buffer (return)
{
    int j = 0; // the counter of #buffers
    buffer.push_back(std::vector<int>());
    buffer[j].reserve(max_buff);
    
    for (int i = 0; i < num_keys; ++i) {
        const int *bin = index + index_pos[i]; // get bin
        int num = index_pos[i+1]-index_pos[i]; // get bin num
        int key = index_key[i];                // get bin key
        
        // using local bin freq to get cand_bin
        int len = 0; // actual length of cand_bin
        int max_freq = frequent_data(num, cand_bin_size, lbeta, bin, bktset, 
            bktpos, tmp_arr, tmp_pid, tmp_freq, cand_bin, len);
        
        if (len == 0) continue; // no data in this bin
        if (len+4 > max_buff) continue;
        
        // add cand_bin to buffer for further processing
        if (buffer[j].size()+len+4 > max_buff) {
            ++j;
            buffer.push_back(std::vector<int>());
            buffer[j].reserve(max_buff);
        }
        buffer[j].push_back(key);
        buffer[j].push_back(num);
        buffer[j].push_back(max_freq);
        buffer[j].push_back(len);
        buffer[j].insert(buffer[j].end(), cand_bin, cand_bin+len);
    }
    return j; // start from 0
}

// -----------------------------------------------------------------------------
struct PairCmp {
__host__ __device__
bool operator() (const std::pair<int,int> &o1, const std::pair<int,int> &o2) {
    return o1.first < o2.first;
}
};

// -----------------------------------------------------------------------------
void frequent_data(                 // get frequent data from cand_list
    int   cand_bin_size,                // total cand_bin size
    float gbeta,                        // global beta
    std::vector<std::pair<int,int> > &cand_list, // cand_list (allow modify)
    int   *tmp_pid,                     // tmp pid list (allow modify)
    int   *tmp_freq,                    // tmp freq list (allow modify)
    int   *cand_bin,                    // cand_bin (return)
    int   &cand_len)                    // actual length of cand_bin (return)
{
    // first sort cand_list in ascending order by PairCmp
    thrust::sort(cand_list.begin(), cand_list.end(), PairCmp());
    
    // get the distinct point IDs and their frequencies
    u64 total_num = cand_list.size();
    if (total_num > STATIC_ARRAY_SIZE) { 
        // printf("total_num = %lu\n", total_num);
        cand_len = 0;
        std::vector<std::pair<int,int> >().swap(cand_list);
        return;
    }
    
    int cnt = cand_list[0].second; // get the frequency of point ID
    int max_freq = -1, num = 0;
    for (size_t i = 1; i < total_num; ++i) {
        if (cand_list[i].first != cand_list[i-1].first) {
            tmp_pid[num] = cand_list[i-1].first; tmp_freq[num] = cnt;
            if (cnt > max_freq) max_freq = cnt;
            
            cnt = cand_list[i].second; ++num;
        }
        else cnt += cand_list[i].second;
    }
    tmp_pid[num] = cand_list[total_num-1].first; tmp_freq[num] = cnt;
    if (cnt > max_freq) max_freq = cnt;
    ++num;
    
    // get the high frequent point IDs & add them to cand_bin
    get_cand_bin(num, max_freq, cand_bin_size, gbeta, tmp_pid, tmp_freq, 
        cand_bin, cand_len);
    
    // release space
    std::vector<std::pair<int,int> >().swap(cand_list);
}

// -----------------------------------------------------------------------------
int signatures_to_bins(             // convert local signatures into global bins
    int   rank,                         // MPI rank
    int   size,                         // number of MPIs (size)
    int   N,                            // total number of data points
    int   n,                            // num of buckets / length of signatures
    int   l,                            // number of minhash tables
    int   b,                            // threshold of #bucket IDs in a bin
    int   delta,                        // threshold of #point  IDs in a bin
    bool  filter,                       // whether filter #point IDs in a bin
    float gbeta,                        // global beta
    float lbeta,                        // local  beta
    const int *bktset,                  // bucket set
    const u64 *bktpos,                  // bucket position
    int   *signatures,                  // minhash signatures (allow modify)
    std::vector<int> &binset,           // bin set (return)
    std::vector<u64> &binpos)           // bin position (return)
{
    // clear binset and binpos
    std::vector<int>().swap(binset);
    std::vector<u64>().swap(binpos);
    
    // estimate cost for binset and binpos
    srand(RANDOM_SEED);         // fix a random seed
    binset.reserve((u64) n*size*MAX_BIN_SIZE); // maybe under-estimate
    binpos.reserve((u64) n*size+1);  // worst case size
    binpos.push_back(0UL);
    
    // -------------------------------------------------------------------------
    //  convert local signatures into global binset and binpos
    // -------------------------------------------------------------------------
    int *tmp_arr   = new int[STATIC_ARRAY_SIZE];
    int *tmp_pid   = new int[STATIC_ARRAY_SIZE];
    int *tmp_freq  = new int[STATIC_ARRAY_SIZE];
    int *cand_bin  = new int[N];
    
    int *index     = new int[n];
    int *index_key = new int[n];   // worst case size
    int *index_pos = new int[n+1]; // worst case size
    
    int *all_buff  = new int[MAX_BUFF_SIZE];
    int  max_buff  = MAX_BUFF_SIZE / size;
    
    for (int i = 0; i < l; ++i) {
        // ---------------------------------------------------------------------
        //  local signature to local bucket index, index_key, and index_pos
        // ---------------------------------------------------------------------
        int num_keys = signature_to_index(n, signatures + (u64) i*n, index, 
            index_key, index_pos);
        
        // ---------------------------------------------------------------------
        //  single-thread case
        // ---------------------------------------------------------------------
        if (size == 1) {
            // convert local bucket index into global bins
            index_to_bins(num_keys, N, b, delta, filter, gbeta, bktset, 
                bktpos, index, index_pos, tmp_arr, tmp_pid, tmp_freq, cand_bin, 
                binset, binpos);
#ifdef DEBUG_INFO
            printf("Rank #%d: Signature %2d/%d, #Keys=%d, #Bins=%d\n", rank, 
                i+1, l, num_keys, binpos.size()-1);
#endif
            continue;
        }
        // ---------------------------------------------------------------------
        //  multi-thread case
        // ---------------------------------------------------------------------
        // step 1: convert local bucket index into local buffer
        std::vector<std::vector<int> > buffer;
        int num_buffer = index_to_buffer(num_keys, N, max_buff, lbeta, bktset,
            bktpos, index, index_key, index_pos, tmp_arr, tmp_pid, tmp_freq, 
            cand_bin, buffer);
        
        // step 2: get max round based on num_buffer
        int max_round = get_max_round(size, num_buffer); // start from 1
        
        // step 3: gather & convert all local buffers into candidate list
        std::unordered_map<int, std::vector<std::pair<int,int> > > cand_list;
        std::unordered_map<int, int> cand_cnt;
        for (int j = 0; j < max_round; ++j) {
            // gather all buffers from different local buffers to root
            if (num_buffer < j) buffer.push_back(std::vector<int>());
            int tlen = gather_all_buffers(size, buffer[j], all_buff);

            // @root: convert all buffers into candidate list
            if (rank == 0) {
                all_buff_to_cand_list(tlen, all_buff, cand_list, cand_cnt);
            }
            std::vector<int>().swap(buffer[j]); // clear buffer[j]
        }
        // step 4: convert candidate list into global bins @root
        if (rank == 0) {
            // get keys in ascending order
            std::vector<int> keys(cand_list.size());
            int j = 0;
            for (auto& p : cand_list) keys[j++] = p.first;
            thrust::sort(keys.begin(), keys.end());
            
            for (int key : keys) {
                if (cand_cnt[key] <= b) continue; // skip if |bin| <= b
                
                // get cand_bin from candidate list
                int len = 0; // actual length of cand_bin
                frequent_data(N, gbeta, cand_list[key], tmp_pid, tmp_freq, 
                    cand_bin, len);
                
                // add cand_bin to binset and binpos
                add_cand_bin(len, delta, filter, cand_bin, binset, binpos);
            }
#ifdef DEBUG_INFO
            printf("Rank #%d: Signature %2d/%d, #Keys=%d, #Bins=%d\n", rank, 
                i+1, l, keys.size(), binpos.size()-1);
#endif
            std::vector<int>().swap(keys);
        }
        MPI_Barrier(MPI_COMM_WORLD);
        
        std::vector<std::vector<int> >().swap(buffer); // clear buffer
        cand_list.clear();
        cand_cnt.clear();
    }
    delete[] tmp_arr;
    delete[] tmp_pid;
    delete[] tmp_freq;
    delete[] cand_bin;
    
    delete[] index;
    delete[] index_key;
    delete[] index_pos;
    delete[] all_buff;
    
    // broadcast global bins from root to other threads
    if (size > 1) broadcast_set_and_pos(rank, size, binset, binpos);
    
    // accumulate the bin size to get the start position of each bin
    size_t n_binpos = binpos.size();
    for (size_t i = 1; i < n_binpos; ++i) binpos[i] += binpos[i-1];
    
    return binpos.size()-1;
}

// -----------------------------------------------------------------------------
u64 labels_to_index(                // convert labels into index and index_pos
    int   size,                         // number of MPIs (size)
    int   n,                            // number of labels
    int   k,                            // number of centers
    const int *labels,                  // cluster labels for data points
    std::vector<int> &binset,           // bin set (return)
    std::vector<u64> &binpos)           // bin position (return)
{
    assert((u64) n*size < MAX_INT);
    
    int N = n*size; // total num of data points
    int *all_labels = new int[N];
    if (size == 1) {
        // directly copy labels to all labels
        std::copy(labels, labels+n, all_labels);
    }
    else {
        // get all labels from different threads to all threads
        MPI_Barrier(MPI_COMM_WORLD);
        MPI_Allgather(labels, n, MPI_INT, all_labels, n, MPI_INT, MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);
    }
    
    // sort all labels and its corresponding index
    int *index = new int[N];
    thrust::sequence(index, index+N); // init index with 0,1,2,3,...
    thrust::sort_by_key(all_labels, all_labels+N, index);
    
    // update binset and binpos
    binset.reserve(N);
    binset.insert(binset.end(), index, index+N);
    
    binpos.reserve(k+1); // reserve by the last number of centers
    binpos.push_back(0UL);
    for (int i = 1; i < N; ++i) {
        if (all_labels[i] != all_labels[i-1]) binpos.push_back(i);
    }
    binpos.push_back(N);
    
    // release space
    delete[] index;
    delete[] all_labels;
    
    return binpos.size()-1;
}

// -----------------------------------------------------------------------------
int labels_to_bins(                 // labels to bins & re-number labels
    int rank,                           // MPI rank
    int size,                           // number of MPIs (size)
    int n,                              // number of data points
    int k,                              // number of cluster centers
    int *labels,                        // cluster labels for data (return)
    std::vector<int> &binset,           // bin set (return)
    std::vector<u64> &binpos)           // bin position (return)
{
    std::vector<int>().swap(binset);
    std::vector<u64>().swap(binpos);
    
    // convert labels on local data into global bin set and bin position
    u64 num_bins = labels_to_index(size, n, k, labels, binset, binpos);
    assert(num_bins < MAX_INT);
    
    // re-number labels for local data
    int lb = n * rank; // lower_bound
    int ub = n + lb;   // upper bound
    for (int i = 0; i < num_bins; ++i) {
        const int *bin = &binset[binpos[i]];    // get bin
        int num = int(binpos[i+1] - binpos[i]); // get bin num
        for (int j = 0; j < num; ++j) {
            int id = bin[j];
            if (id >= lb && id < ub) labels[id-lb] = i;
        }
    }
    return num_bins;
}

} // end namespace clustering
