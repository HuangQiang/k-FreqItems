#include "hip/hip_runtime.h"
#include "assign.cuh"

namespace clustering {

// -----------------------------------------------------------------------------
struct PairCmp {
    __host__ __device__
    bool operator() (const std::pair<int,int> &o1, const std::pair<int,int> &o2)
    {
        return o1.first < o2.first;
    }
};

// -----------------------------------------------------------------------------
void frequent_items(                // apply frequent items for cand_list
    float galpha,                       // global alpha
    std::vector<std::pair<int,int> > &cand_list, // cand_list (allow modify)
    std::vector<int> &seedset,          // seed set (return)
    std::vector<u64> &seedpos)          // seed position (return)
{
    // first sort cand_list in ascending order by PairCmp
    thrust::sort(cand_list.begin(), cand_list.end(), PairCmp());
    
    // get the distinct coordinates and their frequencies
    u64 total_num = cand_list.size();
    int *coord = new int[total_num];
    int *freq  = new int[total_num];
    
    int cnt = cand_list[0].second; // get the frequency of coordinate
    int max_freq = -1, num = 0;
    for (size_t i = 1; i < total_num; ++i) {
        if (cand_list[i].first != cand_list[i-1].first) {
            coord[num] = cand_list[i-1].first; freq[num] = cnt;
            if (cnt > max_freq) max_freq = cnt;
            
            cnt = cand_list[i].second; ++num;
        }
        else cnt += cand_list[i].second;
    }
    coord[num] = cand_list[total_num-1].first; freq[num] = cnt;
    if (cnt > max_freq) max_freq = cnt;
    ++num;
    
    // get the high frequent coord and their freq as seedset and seedpos
    int threshold = (int) ceil((double) max_freq*galpha);
    if (max_freq == 1) {
        // directly use all coords as seed and update seedset & seedpos
        seedpos.push_back(num);
        seedset.insert(seedset.end(), coord, coord+num);
    }
    else {
        // get the high frequent coordinates as seed
        int *seed = new int[num];
        cnt = 0; // number of coordinates for seed
        for (int i = 0; i < num; ++i) {
            if (freq[i] >= threshold) { seed[cnt++] = coord[i]; }
        }
        // update seedset and seedpos
        seedpos.push_back(cnt); // add cnt to seedpos
        seedset.insert(seedset.end(), seed, seed+cnt);
        delete[] seed;
    }
    // release space
    std::vector<std::pair<int,int> >().swap(cand_list);
    delete[] coord;
    delete[] freq;
}

// -----------------------------------------------------------------------------
template<class DType>
__global__ void calc_jaccard_dist(  // calc jaccard distance
    int   batch,                        // batch number of data points
    int   k,                            // number of seeds
    const DType *d_dset,                // data set
    const u64   *d_dpos,                // data position
    const int   *d_sset,                // seed set
    const u64   *d_spos,                // seed position
    u16   *d_dist)                      // jaccard distance (return)
{
    u64 tid = (u64) blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < (u64) batch * k) {
        const u64   *dpos = d_dpos + (u64) tid/k;
        const DType *data = d_dset + dpos[0];
        int n_data = get_length(0, dpos);
        
        const u64 *spos = d_spos + (u64) tid%k;
        const int *seed = d_sset + spos[0];
        int n_seed = get_length(0, spos);
        
        float jaccard = jaccard_dist<DType>(n_data, n_seed, data, seed);
        d_dist[tid] = (u16) ceil(jaccard * 65535);
    }
}

// -----------------------------------------------------------------------------
__global__ void nearest_seed(       // find the nearest seed for batch data
    int   batch,                        // batch number of data points
    int   k,                            // number of seeds
    const u16 *d_dist,                  // jaccard distance array
    int   *d_labels)                    // labels (return)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < batch) {
        const u16 *dist = d_dist + (u64) tid*k; // get dist array
        
        // find the minimum jaccard distance for batch data
        int min_id   = 0;
        u16 min_dist = dist[0];
        for (int i = 1; i < k; ++i) {
            if (dist[i] < min_dist) { min_id = i; min_dist = dist[i]; }
        }
        d_labels[tid] = min_id;
    }
}

// -----------------------------------------------------------------------------
template<class DType>
void approx_assign_data(            // approximate sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const DType *dataset,               // data set
    const u64   *datapos,               // data position
    const int   *seedset,               // seed set
    const u64   *seedpos,               // seed position
    int   *labels)                      // cluster labels for dataset (return)
{
    hipSetDevice(DEVICE_LOCAL_RANK);

    // declare parameters and allocation
    u64 len = seedpos[k];
    int *d_sset; hipMalloc((void**) &d_sset, sizeof(int)*len);
    u64 *d_spos; hipMalloc((void**) &d_spos, sizeof(u64)*(k+1));
    
    hipMemcpy(d_sset, seedset, sizeof(int)*len,   hipMemcpyHostToDevice);
    hipMemcpy(d_spos, seedpos, sizeof(u64)*(k+1), hipMemcpyHostToDevice);

    // mem_avail = total_mem - memory(d_sset + d_spos)
    u64 mem_avail = GPU_MEMORY_LIMIT - (sizeof(int)*len + sizeof(u64)*(k+1));
    u64 mem_usage = 0UL, n_dset = 0UL;
    int n_dpos = 0, batch = 0, start  = 0;
    
    for (int i = 0; i <= n; ++i) {
        // ---------------------------------------------------------------------
        //  calculate memory usage requirement if adding one more data
        // ---------------------------------------------------------------------
        if (i < n) {
            // d_dset + d_dpos + d_dist + d_labels
            n_dset = datapos[i+1] - datapos[start];
            n_dpos = batch + 2;
            mem_usage = sizeof(DType)*n_dset + sizeof(u64)*n_dpos + 
                (sizeof(u16)*k+sizeof(int))*(batch+1);
        }
        // ---------------------------------------------------------------------
        //  parallel batch data assignment if over mem_avail or end
        // ---------------------------------------------------------------------
        if (mem_usage > mem_avail || i == n) {
            n_dset = datapos[i] - datapos[start];
            n_dpos = batch + 1;
            mem_usage = sizeof(DType)*n_dset + sizeof(u64)*n_dpos + 
                (sizeof(u16)*k+sizeof(int))*batch;
#ifdef DEBUG_INFO
            printf("Rank #%d: n=%d, i=%d, batch=%d, mem_usage=%lu, mem_avail=%lu\n",
                rank, n, i, batch, mem_usage, mem_avail);
#endif
            // cuda allocation and memory copy from CPU to GPU
            const DType *h_dset = dataset + datapos[start];
            u64   *h_dpos = new u64[n_dpos];
            copy_pos(n_dpos, datapos + start, h_dpos);
            
            DType *d_dset;   hipMalloc((void**) &d_dset,   sizeof(DType)*n_dset);
            u64   *d_dpos;   hipMalloc((void**) &d_dpos,   sizeof(u64)*n_dpos);
            u16   *d_dist;   hipMalloc((void**) &d_dist,   sizeof(u16)*k*batch);
            int   *d_labels; hipMalloc((void**) &d_labels, sizeof(int)*batch);
            
            hipMemcpy(d_dset, h_dset, sizeof(DType)*n_dset, hipMemcpyHostToDevice);
            hipMemcpy(d_dpos, h_dpos, sizeof(u64)*n_dpos,   hipMemcpyHostToDevice);
            
            // calc Jaccard distance between batch data and k seeds
            int block = BLOCK_SIZE;
            int grid  = ((u64) batch*k + block-1) / block;
            calc_jaccard_dist<DType><<<grid, block>>>(batch, k, d_dset, d_dpos,
                d_sset, d_spos, d_dist);
            
            // find the nearest seed for batch data
            grid = ((u64) batch + block-1) / block;
            nearest_seed<<<grid, block>>>(batch, k, d_dist, d_labels);
            
            // update labels & release local space
            hipMemcpy(labels+start, d_labels, sizeof(int)*batch, hipMemcpyDeviceToHost);
            
            hipFree(d_dset); hipFree(d_dpos);
            hipFree(d_dist); hipFree(d_labels);
            delete[] h_dpos;
            
            // update local parameters for next batch data assignment
            start += batch; batch = 0;
        }
        if (i < n) ++batch;
    }
    assert(start == n);
    hipFree(d_sset); hipFree(d_spos);
}

// -----------------------------------------------------------------------------
template void approx_assign_data(   // approximate sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const u08 *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    int   *labels);                     // cluster labels for data (return)
    
// -----------------------------------------------------------------------------
template void approx_assign_data(   // approximate sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const u16 *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    int   *labels);                     // cluster labels for data (return)
    
// -----------------------------------------------------------------------------
template void approx_assign_data(   // approximate sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const int *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    int   *labels);                     // cluster labels for data (return)
    
// -----------------------------------------------------------------------------
template void approx_assign_data(   // approximate sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const f32 *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    int   *labels);                     // cluster labels for data (return)

// -----------------------------------------------------------------------------
template<class DType>
__global__ void calc_jaccard_dist(  // calc jaccard distance
    int   batch,                        // batch number of data points
    int   k,                            // number of seeds
    const DType *d_dset,                // data set
    const u64   *d_dpos,                // data position
    const int   *d_sset,                // seed set
    const u64   *d_spos,                // seed position
    float *d_dist)                      // jaccard distance (return)
{
    u64 tid = (u64) blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < (u64) batch * k) {
        const u64   *dpos = d_dpos + (u64) tid/k;
        const DType *data = d_dset + dpos[0];
        int n_data = get_length(0, dpos);
        
        const u64 *spos = d_spos + (u64) tid%k;
        const int *seed = d_sset + spos[0];
        int n_seed = get_length(0, spos);
        
        d_dist[tid] = jaccard_dist<DType>(n_data, n_seed, data, seed);
    }
}

// -----------------------------------------------------------------------------
__global__ void nearest_seed(       // find the nearest seed id for batch data
    int   batch,                        // batch number of data points
    int   k,                            // number of seeds
    const float *d_dist,                // jaccard distance array
    int   *d_labels)                    // labels (return)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < batch) {
        const float *dist = d_dist + (u64) tid*k; // get dist array
        
        // find the minimum jaccard distance for batch data
        int   min_id   = 0;
        float min_dist = dist[0];
        for (int i = 1; i < k; ++i) {
            if (dist[i] < min_dist) { min_id = i; min_dist = dist[i]; }
        }
        d_labels[tid] = min_id;
    }
}

// -----------------------------------------------------------------------------
template<class DType>
void exact_assign_data(             // exact sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const DType *dataset,               // data set
    const u64   *datapos,               // data position
    const int   *seedset,               // seed set
    const u64   *seedpos,               // seed position
    int   *labels)                      // cluster labels for dataset (return)
{
    hipSetDevice(DEVICE_LOCAL_RANK);

    // declare parameters and allocation
    u64 len = seedpos[k];
    int *d_sset = nullptr; hipMalloc((void**)&d_sset, sizeof(int)*len);
    u64 *d_spos = nullptr; hipMalloc((void**)&d_spos, sizeof(u64)*(k+1));
    
    hipMemcpy(d_sset, seedset, sizeof(int)*len,   hipMemcpyHostToDevice);
    hipMemcpy(d_spos, seedpos, sizeof(u64)*(k+1), hipMemcpyHostToDevice);

    // mem_avail = total_mem - memory(d_sset + d_spos)
    u64 mem_avail = GPU_MEMORY_LIMIT - (sizeof(int)*len + sizeof(u64)*(k+1));
    u64 mem_usage = 0UL, n_dset = 0UL;
    int n_dpos = 0, batch = 0, start = 0;
    
    for (int i = 0; i <= n; ++i) {
        // ---------------------------------------------------------------------
        //  calculate memory usage requirement if adding one more data
        // ---------------------------------------------------------------------
        if (i < n) {
            // d_dset + d_dpos + d_dist + d_labels
            n_dset = datapos[i+1] - datapos[start];
            n_dpos = batch + 2;
            mem_usage = sizeof(DType)*n_dset + sizeof(u64)*n_dpos + 
                (sizeof(float)*k+sizeof(int))*(batch+1);
        }
        // ---------------------------------------------------------------------
        //  parallel batch data assignment if over mem_avail or end
        // ---------------------------------------------------------------------
        if (mem_usage > mem_avail || i == n) {
            n_dset = datapos[i] - datapos[start];
            n_dpos = batch + 1;
            mem_usage = sizeof(DType)*n_dset + sizeof(u64)*n_dpos + 
                (sizeof(float)*k+sizeof(int))*batch;
#ifdef DEBUG_INFO
            printf("Rank #%d: n=%d, i=%d, batch=%d, mem_usage=%lu, mem_avail=%lu\n",
                rank, n, i, batch, mem_usage, mem_avail);
#endif
            // cuda allocation and memory copy from CPU to GPU
            const DType *h_dset = dataset + datapos[start];
            u64   *h_dpos = new u64[n_dpos];
            copy_pos(n_dpos, datapos + start, h_dpos);
            
            DType *d_dset;   hipMalloc((void**) &d_dset,   sizeof(DType)*n_dset);
            u64   *d_dpos;   hipMalloc((void**) &d_dpos,   sizeof(u64)*n_dpos);
            float *d_dist;   hipMalloc((void**) &d_dist,   sizeof(float)*batch*k);
            int   *d_labels; hipMalloc((void**) &d_labels, sizeof(int)*batch);
            
            hipMemcpy(d_dset, h_dset, sizeof(DType)*n_dset, hipMemcpyHostToDevice);
            hipMemcpy(d_dpos, h_dpos, sizeof(u64)*n_dpos,   hipMemcpyHostToDevice);
            
            // compute Jaccard distance for between batch data and k seeds
            int block = BLOCK_SIZE;
            int grid  = ((u64) batch*k + block-1) / block;
            calc_jaccard_dist<DType><<<grid, block>>>(batch, k, d_dset, d_dpos,
                d_sset, d_spos, d_dist);
            
            // find the nearest seed for batch data
            grid = ((u64) batch + block-1) / block;
            nearest_seed<<<grid, block>>>(batch, k, d_dist, d_labels);
            
            // update labels & release local space
            hipMemcpy(&labels[start], d_labels, sizeof(int)*batch, hipMemcpyDeviceToHost);
            
            hipFree(d_dset); hipFree(d_dpos); 
            hipFree(d_dist); hipFree(d_labels);
            delete[] h_dpos;
            
            // update local parameters for next batch data assignment
            start += batch; batch = 0;
        }
        if (i < n) ++batch;
    }
    assert(start == n);
    hipFree(d_sset); hipFree(d_spos);
}

// -----------------------------------------------------------------------------
template void exact_assign_data(    // exact sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const u08 *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    int   *labels);                     // cluster labels for data (return)
    
// -----------------------------------------------------------------------------
template void exact_assign_data(    // exact sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const u16 *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    int   *labels);                     // cluster labels for data (return)
    
// -----------------------------------------------------------------------------
template void exact_assign_data(    // exact sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const int *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    int   *labels);                     // cluster labels for data (return)
    
// -----------------------------------------------------------------------------
template void exact_assign_data(    // exact sparse data assginment
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const f32 *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    int   *labels);                     // cluster labels for data (return)

} // end namespace clustering
