#include "hip/hip_runtime.h"
#include "seeding.cuh"

namespace clustering {

// -----------------------------------------------------------------------------
void generate_k_distinct_ids(       // generate k distinct ids
    int k,                              // k value
    int n,                              // total range
    int *distinct_ids)                  // distinct ids (return)
{
    bool *select = new bool[n]; memset(select, false, sizeof(bool)*n);
    int  id = -1;
    for (int i = 0; i < k; ++i) {
        // every time draw a distinct id uniformly at from from [0,n-1]
        do { id = uniform_u32(0, n-1); } while (select[id]);
        
        select[id] = true;
        distinct_ids[i] = id;
    }
    delete[] select;
}

// -----------------------------------------------------------------------------
void gather_all_local_seedset(      // gather all local seedsets to root
    int   rank,                         // MPI rank
    int   size,                         // number of MPIs (size)
    const std::vector<int> &local_seedset, // local seedset
    std::vector<int> &seedset)          // seedset at root (return)
{
    // gather the length of local_seedset from different threads to root
    int len = (int) local_seedset.size(); // length of local_seedset
    int *rlen = new int[size];
    for (int i = 0; i < size; ++i) rlen[i] = 0;
    
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Gather(&len, 1, MPI_INT, rlen, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    // @root: get the total length of global seedset & resize it
    int tlen = 0;
    for (int i = 0; i < size; ++i) tlen += rlen[i];
    if (rank == 0) { assert(tlen <= MAX_BUFF_SIZE); seedset.resize(tlen); }
    
    // @root: init displacements to gather all local_seedset
    int *displs = new int[size];
    displs[0] = 0;
    for (int i = 1; i < size; ++i) displs[i] = displs[i-1] + rlen[i-1];

    // gather local_seedset from different threads to global seedset @root
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Gatherv(local_seedset.data(), len, MPI_INT, seedset.data(), rlen, 
        displs, MPI_INT, 0, MPI_COMM_WORLD); // 0 is root location
    MPI_Barrier(MPI_COMM_WORLD);
    
    delete[] rlen;
    delete[] displs;
}

// -----------------------------------------------------------------------------
void gather_all_local_seedpos(      // gather all local seedpos to root
    int   rank,                         // MPI rank
    int   size,                         // number of MPIs (size)
    int   k,                            // k value
    const std::vector<u64> &local_seedpos, // local seedpos
    std::vector<u64> &seedpos)          // seedpos at root (return)
{
    // gather the length of local_seedpos from different threads to root
    int len = (int) local_seedpos.size()-1; // skip the first 0
    int *rlen = new int[size];
    for (int i = 0; i < size; ++i) rlen[i] = 0;
    
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Gather(&len, 1, MPI_INT, rlen, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    // @root: get the total length of global seedpos & reinit it
    int tlen = 0;
    for (int i = 0; i < size; ++i) tlen += rlen[i];
    if (rank == 0) { assert(tlen == k); seedpos.resize(k+1); seedpos[0]=0; }
    
    // @root: init displacements to gather all local_seedpos
    int *displs = new int[size];
    displs[0] = 0;
    for (int i = 1; i < size; ++i) displs[i] = displs[i-1] + rlen[i-1];

    // gather local_seedpos from different threads to global seedpos @root
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Gatherv(local_seedpos.data()+1, len, MPI_UINT64_T, seedpos.data()+1, 
        rlen, displs, MPI_UINT64_T, 0, MPI_COMM_WORLD); // 0 is root location
    MPI_Barrier(MPI_COMM_WORLD);
    
    delete[] rlen;
    delete[] displs;
}

// -----------------------------------------------------------------------------
template<class DType>
__global__ void calc_jaccard_dist(  // calc jaccard distance
    int   batch,                        // batch number of data points
    int   n_seed,                       // length of seed
    const int   *d_seed,                // seed
    const DType *d_dset,                // data set
    const u64   *d_dpos,                // data position
    float *d_dist)                      // jaccard distance (return)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < batch) {
        const u64   *dpos = &d_dpos[tid];
        const DType *data = &d_dset[dpos[0]];
        int n_data = get_length(0, dpos);
        
        float dist = jaccard_dist<DType>(n_data, n_seed, data, d_seed);
        if (d_dist[tid] > dist) d_dist[tid] = dist;
    }
}

// -----------------------------------------------------------------------------
template<class DType>
void update_nn_dist(                // update nn_dist for local data
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    const std::vector<int> &seed,       // last seed
    const DType *dataset,               // data set
    const u64   *datapos,               // data position
    float *nn_dist)                     // nn_dist (return)
{
    hipSetDevice(DEVICE_LOCAL_RANK);
    
    // declare parameters and allocation
    int n_seed  = (int) seed.size();
    int *d_seed = nullptr; hipMalloc((void**)&d_seed, sizeof(int)*n_seed);
    hipMemcpy(d_seed, seed.data(), sizeof(int)*n_seed, hipMemcpyHostToDevice);
    
    // mem_avail = total_mem - memory(d_seed)
    u64 mem_avail = GPU_MEMORY_LIMIT - sizeof(int)*n_seed;
    u64 mem_usage = 0UL, n_dset = 0UL;
    int n_dpos = 0, batch = 0, start  = 0;
    
    for (int i = 0; i <= n; ++i) {
        // ---------------------------------------------------------------------
        //  calculate memory usage requirement if adding one more data
        // ---------------------------------------------------------------------
        if (i < n) {
            // d_dset + d_dpos + d_dist
            n_dset = datapos[i+1] - datapos[start];
            n_dpos = batch + 2;
            mem_usage = sizeof(DType)*n_dset + sizeof(u64)*n_dpos + 
                sizeof(float)*(batch+1);
        }
        // ---------------------------------------------------------------------
        //  parallel nn_dist update for batch data if over mem_avail or end
        // ---------------------------------------------------------------------
        if (mem_usage > mem_avail || i == n) {
            n_dset = datapos[i] - datapos[start];
            n_dpos = batch + 1;
            mem_usage = sizeof(DType)*n_dset + sizeof(u64)*n_dpos + 
                sizeof(float)*batch;
#ifdef DEBUG_INFO
            printf("Rank #%d: n=%d, i=%d, batch=%d, mem_usage=%lu, mem_avail=%lu\n",
                rank, n, i, batch, mem_usage, mem_avail);
#endif
            // cuda allocation and memory copy from CPU to GPU 
            const DType *h_dset = dataset + datapos[start];
            float *h_dist = nn_dist + start; // allow modify
            u64   *h_dpos = new u64[n_dpos];
            copy_pos(n_dpos, datapos + start, h_dpos);
            
            DType *d_dset; hipMalloc((void**) &d_dset, sizeof(DType)*n_dset);
            u64   *d_dpos; hipMalloc((void**) &d_dpos, sizeof(u64)*n_dpos);
            float *d_dist; hipMalloc((void**) &d_dist, sizeof(float)*batch);
            
            hipMemcpy(d_dset, h_dset, sizeof(DType)*n_dset, hipMemcpyHostToDevice);
            hipMemcpy(d_dpos, h_dpos, sizeof(u64)*n_dpos,   hipMemcpyHostToDevice);
            hipMemcpy(d_dist, h_dist, sizeof(float)*batch,  hipMemcpyHostToDevice);
            
            // calc Jaccard distance for between batch data and seed
            int block = BLOCK_SIZE;
            int grid  = ((u64) batch + block-1) / block;
            calc_jaccard_dist<DType><<<grid, block>>>(batch, n_seed, d_seed, 
                d_dset, d_dpos, d_dist);
            
            // update nn_dist for batch data & release local space
            hipMemcpy(h_dist, d_dist, sizeof(float)*batch, hipMemcpyDeviceToHost);
            
            hipFree(d_dset); hipFree(d_dpos); hipFree(d_dist);
            delete[] h_dpos;
            
            // update local parameters for next nn_dist update
            start += batch; batch = 0;
        }
        if (i < n) ++batch;
    }
    assert(start == n);
    hipFree(d_seed);
}

// -----------------------------------------------------------------------------
template void update_nn_dist(       // update nn_dist for local data
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    const std::vector<int> &seed,       // last seed
    const u08 *dataset,                 // data set
    const u64 *datapos,                 // data position
    float *nn_dist);                    // nn_dist (return)

// -----------------------------------------------------------------------------
template void update_nn_dist(       // update nn_dist for local data
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    const std::vector<int> &seed,       // last seed
    const u16 *dataset,                 // data set
    const u64 *datapos,                 // data position
    float *nn_dist);                    // nn_dist (return)

// -----------------------------------------------------------------------------
template void update_nn_dist(       // update nn_dist for local data
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    const std::vector<int> &seed,       // last seed
    const int *dataset,                 // data set
    const u64 *datapos,                 // data position
    float *nn_dist);                    // nn_dist (return)

// -----------------------------------------------------------------------------
template void update_nn_dist(       // update nn_dist for local data
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    const std::vector<int> &seed,       // last seed
    const f32 *dataset,                 // data set
    const u64 *datapos,                 // data position
    float *nn_dist);                    // nn_dist (return)

// -----------------------------------------------------------------------------
void broadcast_target_data(         // broadcast target data to all threads
    int   rank,                         // MPI rank
    int   size,                         // number of MPIs (size)
    std::vector<int> &target_data)      // target data (return)
{
    // gather the length of target_data from different threads to root (rank=0)
    int len = (int) target_data.size(); // length of target_data
    int *rlen = new int[size];
    for (int i = 0; i < size; ++i) rlen[i] = 0;
    
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Gather(&len, 1, MPI_INT, rlen, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    // @root (rank=0): determine the rank of local_data as new_root
    int new_root = 0;
    for (int i = 0; i < size; ++i) if (rlen[i] > 0) new_root = i;
    
    // broadcast the new_root from old root (rank=0) to all threads
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(&new_root, 1, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    // broadcast the len and the target_data from new_root to all threads
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(&len, 1, MPI_INT, new_root, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    if (rank != new_root) target_data.resize(len);
    
    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Bcast(target_data.data(), len, MPI_INT, new_root, MPI_COMM_WORLD);
    MPI_Barrier(MPI_COMM_WORLD);
    
    delete[] rlen;
}

// -----------------------------------------------------------------------------
template<class DType>
__global__ void calc_jaccard_dist(  // calc jaccard distance
    int   batch,                        // batch number of data points
    int   k,                            // number of seeds
    const DType *d_dset,                // data set
    const u64   *d_dpos,                // data position
    const int   *d_sset,                // seed set
    const u64   *d_spos,                // seed position
    float *d_dist)                      // jaccard distance (return)
{
    u64 tid = (u64) blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < (u64) batch * k) {
        const u64   *dpos = d_dpos + (u64) tid/k;
        const DType *data = d_dset + dpos[0];
        int n_data = get_length(0, dpos);
        
        const u64 *spos = d_spos + (u64) tid%k;
        const int *seed = d_sset + spos[0];
        int n_seed = get_length(0, spos);
        
        d_dist[tid] = jaccard_dist<DType>(n_data, n_seed, data, seed);
    }
}

// -----------------------------------------------------------------------------
__global__ void update_nn_dist(     // update nn_dist for batch data
    int   batch,                        // batch number of data points
    int   k,                            // number of seeds
    const float *d_dist,                // jaccard distance array
    float *d_nn_dist)                   // nn_dist (return)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < batch) {
        const float *dist = d_dist + (u64) tid*k; // get dist array
        float min_dist = d_nn_dist[tid];
        
        // update min_dist among the k dist
        for (int i = 0; i < k; ++i) {
            if (dist[i] < min_dist) min_dist = dist[i];
        }
        d_nn_dist[tid] = min_dist;
    }
}

// -----------------------------------------------------------------------------
template<class DType>
void update_nn_dist(                // update nn_dist for local data by k seeds
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const DType *dataset,               // data set
    const u64   *datapos,               // data position
    const int   *seedset,               // seed set
    const u64   *seedpos,               // seed position
    float *nn_dist)                     // nn_dist (return)
{
    hipSetDevice(DEVICE_LOCAL_RANK);

    // declare parameters and allocation
    u64 len = seedpos[k];
    int *d_sset = nullptr; hipMalloc((void**)&d_sset, sizeof(int)*len);
    u64 *d_spos = nullptr; hipMalloc((void**)&d_spos, sizeof(u64)*(k+1));
    
    hipMemcpy(d_sset, seedset, sizeof(int)*len,   hipMemcpyHostToDevice);
    hipMemcpy(d_spos, seedpos, sizeof(u64)*(k+1), hipMemcpyHostToDevice);

    // mem_avail = total_mem - memory(d_sset + d_spos)
    u64 mem_avail = GPU_MEMORY_LIMIT - (sizeof(int)*len + sizeof(u64)*(k+1));
    u64 mem_usage = 0UL, n_dset = 0UL;
    int n_dpos = 0, batch = 0, start  = 0;
    
    for (int i = 0; i <= n; ++i) {
        // assert(batch == i-start);
        // ---------------------------------------------------------------------
        //  calculate memory usage requirement if adding one more data
        // ---------------------------------------------------------------------
        if (i < n) {
            // d_dset + d_dpos + d_dist + d_nn_dist
            n_dset = datapos[i+1] - datapos[start];
            n_dpos = batch + 2;
            mem_usage = sizeof(DType)*n_dset + sizeof(u64)*n_dpos + 
                sizeof(float)*(k+1)*(batch+1);
        }
        // ---------------------------------------------------------------------
        //  parallel batch data assignment if over mem_avail or end
        // ---------------------------------------------------------------------
        if (mem_usage > mem_avail || i == n) {
            n_dset = datapos[i] - datapos[start];
            n_dpos = batch + 1;
            mem_usage = sizeof(DType)*n_dset + sizeof(u64)*n_dpos + 
                sizeof(float)*(k+1)*batch;
#ifdef DEBUG_INFO
            printf("Rank #%d: n=%d, i=%d, batch=%d, mem_usage=%lu, mem_avail=%lu\n",
                rank, n, i, batch, mem_usage, mem_avail);
#endif
            // cuda allocation and memory copy from CPU to GPU
            const DType *h_dset = dataset + datapos[start];
            float *h_nn_dist = nn_dist + start; // allow modify
            u64   *h_dpos = new u64[n_dpos];
            copy_pos(n_dpos, datapos + start, h_dpos);
            
            DType *d_dset;    hipMalloc((void**) &d_dset,    sizeof(DType)*n_dset);
            u64   *d_dpos;    hipMalloc((void**) &d_dpos,    sizeof(u64)*n_dpos);
            float *d_dist;    hipMalloc((void**) &d_dist,    sizeof(float)*batch*k);
            float *d_nn_dist; hipMalloc((void**) &d_nn_dist, sizeof(float)*batch);
            
            hipMemcpy(d_dset,    h_dset,    sizeof(DType)*n_dset, hipMemcpyHostToDevice);
            hipMemcpy(d_dpos,    h_dpos,    sizeof(u64)*n_dpos,   hipMemcpyHostToDevice);
            hipMemcpy(d_nn_dist, h_nn_dist, sizeof(float)*batch,  hipMemcpyHostToDevice);
            
            // calc Jaccard distance between batch data and k seeds
            int block = BLOCK_SIZE;
            int grid  = ((u64) batch*k + block-1) / block;
            calc_jaccard_dist<DType><<<grid, block>>>(batch, k, d_dset, d_dpos,
                d_sset, d_spos, d_dist);
            
            // update the nn_dist for batch data
            grid = ((u64) batch + block-1) / block;
            update_nn_dist<<<grid, block>>>(batch, k, d_dist, d_nn_dist);
            
            // get the new nn_dist & release local space
            hipMemcpy(h_nn_dist, d_nn_dist, sizeof(float)*batch, hipMemcpyDeviceToHost);
            
            hipFree(d_dset); hipFree(d_dpos);
            hipFree(d_dist); hipFree(d_nn_dist);
            delete[] h_dpos;
            
            // update local parameters for next batch data assignment
            start += batch; batch = 0;
        }
        if (i < n) ++batch;
    }
    assert(start == n);
    hipFree(d_sset); hipFree(d_spos);
}

// -----------------------------------------------------------------------------
template void update_nn_dist(       // update nn_dist for local data by k seeds
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const u08 *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    float *nn_dist);                    // nn_dist (return)
    
// -----------------------------------------------------------------------------
template void update_nn_dist(       // update nn_dist for local data by k seeds
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const u16 *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    float *nn_dist);                    // nn_dist (return)
    
// -----------------------------------------------------------------------------
template void update_nn_dist(       // update nn_dist for local data by k seeds
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const int *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    float *nn_dist);                    // nn_dist (return)
    
// -----------------------------------------------------------------------------
template void update_nn_dist(       // update nn_dist for local data by k seeds
    int   rank,                         // MPI rank
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const f32 *dataset,                 // data set
    const u64 *datapos,                 // data position
    const int *seedset,                 // seed set
    const u64 *seedpos,                 // seed position
    float *nn_dist);                    // nn_dist (return)

// -----------------------------------------------------------------------------
void labels_to_weights(             // convert local labels to global weights
    int   rank,                         // MPI rank
    int   size,                         // number of MPIs (size)
    int   n,                            // number of data points
    int   k,                            // number of seeds
    const int *labels,                  // labels for n local data
    int   *weights)                     // weights for k seeds (return)
{
    assert((u64) n*size < MAX_INT); // total num of data points
    int N = n*size; 
    int *all_labels = new int[N];
    
    // -------------------------------------------------------------------------
    //  get all labels from different threads to root
    // -------------------------------------------------------------------------
    if (size == 1) {
        // directly copy labels to all labels
        std::copy(labels, labels + n, all_labels);
    }
    else {
        // get all labels from different threads to root
        MPI_Barrier(MPI_COMM_WORLD);
        MPI_Gather(labels, n, MPI_INT, all_labels, n, MPI_INT, 0, MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);
    }
    
    // -------------------------------------------------------------------------
    //  @root: sequentical counting the number of labels as weight for each seed
    // -------------------------------------------------------------------------
    if (rank == 0) {
        memset(weights, 0, sizeof(int)*k); // init weights
        int pos = -1;
        for (int i = 0; i < N; ++i) { pos = all_labels[i]; ++weights[pos]; }
    }
    delete[] all_labels;
}

// -----------------------------------------------------------------------------
int early_stop(                     // early stop process
    std::vector<int> &over_seedset,     // over seed set (allow modify)
    std::vector<u64> &over_seedpos,     // over seed position (allow modify)
    std::vector<int> &seedset,          // seed set (return)
    std::vector<u64> &seedpos)          // seed position (return)
{
    // clear original space for seedset & seedpos
    std::vector<int>().swap(seedset);
    std::vector<u64>().swap(seedpos);
    
    // swap contents for over_seedset and seedset
    seedset.swap(over_seedset);
    seedpos.swap(over_seedpos);
    
    return seedpos.size()-1;
}

} // end namespace clustering
